#include "hip/hip_runtime.h"
/*
    GPU-accelerated Thermodynamic Topology Optimization
*/

#include <iostream>
#include <vector>
#include "include/cudakernels.h"
#include "include/assemble.h"
#include "include/solver.h"
#include "include/tdo.h"
#include "include/vtk.h"
#include <iomanip>
#include <ctime>

using namespace std;


// TODO: store local k matrix in constant/texture memory
// TODO: __device__ valueAt() has x and y mixed up
// NOTE:CHECK: when using shared memory, more than one block, get this error : CUDA error for hipMemcpy( ...)
// TODO: check that all kernels have (row, col) formats
// TODO: h = diagonal length of the quad
// TODO: have cudamemcpy for prol and rest matrices to be outside of the function
// TODO: deallocation
// TODO: RAP_ use shared?

// URGENT !!!
// TODO: TODO: size_t is used instead of int in matrix assembly (fillProl ...), shouldn't be used as size_t can't contain negative values!!

//// PARALELLIZABLE / OPTIMIZATION
// TODO: fillIndexVector_GPU()
// TODO: shared memory, use 8 bytes for double precision to avoid bank conflict
            // hipDeviceSetSharedMemConfig( hipSharedMemBankSizeEightByte )
            // see notes in compendium

//// LOW PRIORITY
// TODO: VTK class
// TODO: RA and AP's valueAt(indices) are a bit messed up and confusing
// TODO: 3d assembly, add for loop
// TODO: tried to fix ApplyTranspose for r_coarse to Apply using R-matrix, but didn't work. using ApplyTranspose for now
    // not much time difference though, from 0.01 to 0.007
    

int main()
{

    // create vtk files
    bool writeToVTK = true;

    // output compliance, stiffness and MOD
    bool CSMOD = true;

    // material properties
    double youngMod = 210000;
    double poisson = 0.33;

    //// model set-up
    size_t numLevels = 10;
    
    vector<size_t> N;
    vector<vector<size_t>> bc_index(numLevels);

    size_t update_steps = 100;
    double c_tol = 1e-4;
    bool gmg_verbose = 0;
    bool pcg_verbose = 0;
    bool gmg_verbose_ = 0;
    bool pcg_verbose_ = 0;


    // CASE 0 : 2D MBB
    N = {3,1};                  // domain dimension (x,y,z) on coarsest grid
    double h_coarse = 1;        // local element mesh size on coarsest grid
    size_t bc_case = 0;
    double damp = 2.0/3.0;      // smoother (jacobi damping parameter)


    // // CASE 1 : 3D MBB
    // N = {6,2,1};                // domain dimension (x,y,z) on coarsest grid
    // double h_coarse = 0.5;      // local element mesh size on coarsest grid
    // size_t bc_case = 1;
    // double damp = 1.0/3.0;      // smoother (jacobi damping parameter)

    
    // applying boundary conditions
    size_t dim = N.size();
    bc_index = applyBC(N, numLevels, bc_case, dim);

    // calculating the mesh size on the top level grid
    double h = h_coarse/pow(2,numLevels - 1);
    size_t local_num_rows = pow(2,dim)*dim;

    // TDO
    double rho = 0.3;
    size_t p = 3;
    double etastar = 12.0;
    double betastar = 2 * pow(h,2);

    vector<size_t> num_rows;
    vector<size_t> max_row_size;
    vector<size_t> p_max_row_size;
    vector<size_t> r_max_row_size;

    //// device pointers
    // local stiffness
    double* d_A_local;

    // global stiffness matrix on each grid-level
    vector<double*> d_value;
    vector<size_t*> d_index;

    // prolongation matrices
    vector<double*> d_p_value;
    vector<size_t*> d_p_index;


    // design variable
    double* d_chi;

    //// CUDA
    vector<size_t*> d_node_index;
    size_t* d_node_index_;

    //// benchmarking stuff
    // cuda event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    // creating output file
    // obtaining time and date for output filename
    time_t now = time(0);
    tm *ltm = localtime(&now);
    
    string fileformat_(".txt");
    stringstream ssbm; 
    ssbm << "outputs/" << 1900 + ltm->tm_year << "_" << 1 + ltm->tm_mon << "_" << ltm->tm_mday << "_" << 2 + ltm->tm_hour << ltm->tm_min << 1 + ltm->tm_sec << "_" << dim << "d_lvl_" << numLevels;
    ssbm << fileformat_;
    ofstream ofssbm(ssbm.str(), ios::out);


    // NOTE: insert note here to include in the output file:
    ofssbm << "note: " << endl;

    ofssbm << "### GPU-accelerated Thermodynamic Topology Optimization ###" << endl;
    ofssbm << "Dimension: " << dim << endl;
    ofssbm << "Number of Multigrid Levels: " << numLevels << endl;

    


    // for overall benchmark
    hipEvent_t start_, stop_;
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
    float milliseconds_;
    hipEventRecord(start_);

    /* ##################################################################
    #                           ASSEMBLY                                #
    ###################################################################*/
       

    Assembler Assembly(dim, h, N, youngMod, poisson, rho, p, numLevels);
    Assembly.setBC(bc_index);

        hipEventRecord(start);
    Assembly.init_GPU(d_A_local, d_value, d_index, d_p_value, d_p_index, d_chi, num_rows, max_row_size, p_max_row_size, r_max_row_size, d_node_index, d_node_index_, ofssbm);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        ofssbm << endl;
        ofssbm << "Total assembly time\t\t" << milliseconds << endl;



    cout << "### GPU-accelerated Thermodynamic Topology Optimization ###" << endl;
    cout << "Dimension: " << dim << endl;
    cout << "Number of Multigrid Levels: " << numLevels << endl;
    cout << "Top-level grid size = { " << Assembly.getGridSize()[0];
   
        for ( int i = 1 ; i < dim ; ++i )
            cout << ", " << Assembly.getGridSize()[i];
        
    cout << " }" << endl;
    
    cout << "Top-level mesh size = " << h << endl;
    cout << "Top-level number of rows = " << num_rows[numLevels - 1] << endl;
    cout << "Number of Elements = " << Assembly.getNumElements() << endl;
    cout << "Assembly ... DONE" << endl;
  
    // load vector, b
    vector<double> b(num_rows[numLevels - 1], 0);
    double force = -1;
    applyLoad(b, N, numLevels, bc_case, dim, force);
   

    double* d_u;
    double* d_b;
    CUDA_CALL( hipMalloc((void**)&d_u, sizeof(double) * num_rows[numLevels - 1] ) );
    CUDA_CALL( hipMalloc((void**)&d_b, sizeof(double) * num_rows[numLevels - 1] ) );

    CUDA_CALL( hipMemset(d_u, 0, sizeof(double) * num_rows[numLevels - 1]) );
    CUDA_CALL( hipMemcpy(d_b, &b[0], sizeof(double) * num_rows[numLevels - 1], hipMemcpyHostToDevice) );


    /* ##################################################################
    #                           SOLVER                                  #
    ###################################################################*/

    Solver GMG(d_value, d_index, max_row_size, d_p_value, d_p_index, p_max_row_size, numLevels, num_rows, damp);
    
    
    GMG.set_convergence_params(100000, 1e-15, 1e-8);
    GMG.set_bs_convergence_params(1000, 1e-15, 1e-7);
    GMG.init();
    GMG.set_verbose(gmg_verbose,pcg_verbose);
    GMG.set_num_prepostsmooth(3,3);
    GMG.set_cycle('V');
            
                ofssbm << endl;
                ofssbm << "SOLVER" << endl;
                hipEventRecord(start);
    GMG.solve(d_u, d_b, d_value, ofssbm);
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);
                ofssbm << endl;
                ofssbm << "Total solver time\t\t" << milliseconds << endl;
            

    cout << "Solver   ... DONE" << endl;

    /* ##################################################################
    #                         DENSITY UPDATE                            #
    ###################################################################*/

    // structure compliance
    double c = 0;
    double last_c = 0;
    double MOD;
    double* d_c;
    double* d_MOD;
    CUDA_CALL( hipMalloc((void**)&d_c, sizeof(double) ) );
    CUDA_CALL( hipMalloc((void**)&d_MOD, sizeof(double) ) );


    TDO tdo(d_u, d_chi, h, dim, betastar, etastar, Assembly.getNumElements(), local_num_rows, d_A_local, d_node_index, Assembly.getGridSize(), rho, numLevels, p, d_node_index_);
    tdo.init();
    tdo.set_verbose(0);
                ofssbm << endl;
                ofssbm << "DENSITY UPDATE" << endl;
                hipEventRecord(start);
    tdo.innerloop(d_u, d_chi, d_c, d_MOD, ofssbm);    // get updated d_chi
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);
                ofssbm << endl;
                ofssbm << "Total density update time\t" << milliseconds << endl;
    tdo.print_VTK(0);
    last_c = c;
    CUDA_CALL( hipMemcpy(&c, d_c, sizeof(double), hipMemcpyDeviceToHost) );
    CUDA_CALL( hipMemcpy(&MOD, d_MOD, sizeof(double), hipMemcpyDeviceToHost) );

    // vtk
    vector<double> chi(Assembly.getNumElements(), rho);
    vector<double> u(Assembly.getNumNodes() * dim, 0);
    string fileformat(".vtk");
    int file_index = 0;
    stringstream ss; 
    ss << "vtk/tdo";
    ss << file_index;
    ss << fileformat;

    if ( writeToVTK )
    {
        WriteVectorToVTK(chi, u, ss.str(), dim, Assembly.getNumNodesPerDim(), h, Assembly.getNumElements(), Assembly.getNumNodes() );
        
        CUDA_CALL( hipMemcpy(&chi[0], d_chi, sizeof(double) * Assembly.getNumElements(), hipMemcpyDeviceToHost) );
        CUDA_CALL( hipMemcpy(&u[0], d_u, sizeof(double) * u.size(), hipMemcpyDeviceToHost) );

        file_index++;
        ss.str( string() );
        ss.clear();
        ss << "vtk/tdo";
        ss << file_index;
        ss << fileformat;
        
        WriteVectorToVTK(chi, u, ss.str(), dim, Assembly.getNumNodesPerDim(), h, Assembly.getNumElements(), Assembly.getNumNodes() );
    }
    
    
    size_t iterations = 1;
    float sum_a = 0;  // sum of assembly time
    float sum_s = 0;  // sum of solver time
    float sum_du = 0; // sum of density update time
    float c_rel = abs( c - last_c) / c;
    float init_E = c * pow(rho,p) * youngMod;
    
    ofssbm << endl;
    ofssbm << "   Assembly		Solver				    Density update\t\tCompliance\tStiffness\tMOD\t\tElapsed Time" << endl;
    ofssbm << "   (total)         (total, average, no_iter)    (total, total_bi, no_iter, avg_bi)" << endl;


    hipEventRecord(stop_);
    hipEventSynchronize(stop_);
    milliseconds_ = 0;
    hipEventElapsedTime(&milliseconds_, start_, stop_);
    float elapsed_time = milliseconds_;


    
    // for ( int i = 1 ; i < (update_steps) && ( c_rel > c_tol ); ++i )
    for ( int i = 1 ; i < (update_steps); ++i )
    {
        hipEventRecord(start_);
        ofssbm << setw(4) << left << iterations;
        // update the global stiffness matrix with the updated density distribution
        hipEventRecord(start);
        Assembly.UpdateGlobalStiffness(d_chi, d_value, d_index, d_p_value, d_p_index, d_A_local);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        ofssbm << setw(16) << left << milliseconds;
        sum_a += milliseconds;


        cout << "Calculating iteration " << i << " ... " << endl;
        // hipDeviceSynchronize();
        hipEventRecord(start);
        GMG.reinit();
        GMG.setBM(true);
        GMG.set_convergence_params(1500000, 1e-99, 1e-10);
        GMG.set_bs_convergence_params(1000, 1e-99, 1e-13);
        GMG.set_verbose(gmg_verbose_, pcg_verbose_);
        GMG.solve(d_u, d_b, d_value, ofssbm);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        ofssbm << setw(7) << left << milliseconds << ", " << setw(7) << left << milliseconds/GMG.getCounter() << ", " << GMG.getCounter() << "\t";
        sum_s += milliseconds;

        cout << "Solver done ... " << endl;
        hipEventRecord(start);
        tdo.setBM(true);
        tdo.innerloop(d_u, d_chi, d_c, d_MOD, ofssbm);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        ofssbm << setw(7) << left << milliseconds << ", " << setw(7) << left << tdo.getSum() << ", " << tdo.getCounter() << ", " << tdo.getSum()/tdo.getCounter();
        sum_du += milliseconds;
        
        // collect data of compliance, stiffness and MOD
        if ( CSMOD )
        {
            // convergence check with compliance
            last_c = c;
            CUDA_CALL( hipMemcpy(&c, d_c, sizeof(double), hipMemcpyDeviceToHost) );
            c_rel = abs( c - last_c ) / c;
            ofssbm << "\t\t" << c;

            // structural stiffness, E = 1 / c * E_init
            ofssbm << "\t" << init_E / c;
            
            // computing MOD
            CUDA_CALL( hipMemcpy(&MOD, d_MOD, sizeof(double), hipMemcpyDeviceToHost) );
            ofssbm << "\t\t" << setw(8) << left << MOD;
        }

        cout << "Density update done ... " << endl;

        if ( writeToVTK )
        { 
            CUDA_CALL( hipMemcpy(&chi[0], d_chi, sizeof(double) * Assembly.getNumElements(), hipMemcpyDeviceToHost) );
            CUDA_CALL( hipMemcpy(&u[0], d_u, sizeof(double) * u.size(), hipMemcpyDeviceToHost) );

            file_index++;
            ss.str( string() );
            ss.clear();
            ss << "vtk/tdo";
            ss << file_index;
            ss << fileformat;
            
            WriteVectorToVTK(chi, u, ss.str(), dim, Assembly.getNumNodesPerDim(), h, Assembly.getNumElements(), Assembly.getNumNodes() );

        }        

        hipEventRecord(stop_);
        hipEventSynchronize(stop_);
        milliseconds_ = 0;
        hipEventElapsedTime(&milliseconds_, start_, stop_);
        elapsed_time += milliseconds_;
        ofssbm << "\t" << elapsed_time << endl;


        iterations++;
        
    }

    // hipEventRecord(stop_);
    // hipEventSynchronize(stop_);
    // milliseconds_ = 0;
    // hipEventElapsedTime(&milliseconds_, start_, stop_);
    ofssbm << endl;
    ofssbm << "Average assembly time\t\t" << sum_a / (iterations - 1) << endl;
    ofssbm << "Average solver time\t\t" << sum_s / (iterations - 1) << endl;
    ofssbm << "Average density update time\t" << sum_du / (iterations - 1)  << endl;
    ofssbm << endl;
    ofssbm << "Number of TDO iterations\t" << (iterations - 1) << endl;
    ofssbm << "Time per TDO step \t\t" << elapsed_time/(iterations - 1) << endl;
    ofssbm << "TOTAL RUNTIME\t\t\t" << elapsed_time << endl;
   
    hipDeviceSynchronize();
    
}

/*
    
*/

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime.h>
// #include "../include/mycudaheader.h"
// #include "precond.h"
#include "cudakernels.h"
#include "assemble.h"
#include "solver.h"
#include "tdo.h"

using namespace std;


int main()
{
    // Material properties
    double youngMod = 210e6;
    double poisson = 0.3;

    // domain dimensions
    size_t dim = 2;
    double h = 0.5;     // local element size

    // number of elements per dimension
    // size_t Nx = 1;
    // size_t Ny = 1;

    vector<size_t> N = {1,1};

    // multigrid precond
    size_t numLevels = 2;

    // smoother (jacobi damping parameter)
    double damp = 2.0/3.0;
    
    // boundary conditions
    vector<size_t> bc_index = {0, 1, 6, 7, 12, 13};

    // TDO
    double rho = 0.4;
    size_t p = 3;

    vector<size_t> num_rows;
    vector<size_t> max_row_size;
    vector<size_t> p_max_row_size;

    //// device pointers

    // local stiffness
    double* d_A_local;
    // global stiffness matrix on each grid-level
    vector<double*> d_value;
    vector<size_t*> d_index;

    // prolongation matrices
    vector<double*> d_p_value;
    vector<size_t*> d_p_index;

    // design variable
    double* d_kai;           // NOTE: can alloc this immediately

    // vector u, b
    vector<double> b(18, 0);
    b[5] = -10000;
    double* d_u;
    double* d_b;

    //// CUDA
    vector<size_t*> d_node_index;
    // d_node_index.resize(4);


    // TODO: get num_rows
    CUDA_CALL( hipMalloc((void**)&d_u, sizeof(double) * 18 ) );
    CUDA_CALL( hipMalloc((void**)&d_b, sizeof(double) * 18 ) );

    CUDA_CALL( hipMemset(d_u, 0, sizeof(double) * 18) );

    CUDA_CALL( hipMemcpy(d_b, &b[0], sizeof(double) * 18, hipMemcpyHostToDevice) );
    
    Assembler Assembly(dim, h, N, youngMod, poisson, rho, p, numLevels);
    Assembly.setBC(bc_index);
    Assembly.init(d_A_local, d_value, d_index, d_p_value, d_p_index, d_kai, num_rows, max_row_size, p_max_row_size, d_node_index);

    /*
    NOTE: after assembling you should have these :
    global stiffness matrix ELLPACK
        - vector<double*> d_value(numLevels)
        - vector<size_t> d_index(numLevels)
        - vector<size_t> max_row_size(numLevels)
        - vector<double*> d_p_value(numLevels - 1)
        - vector<size_t*> d_p_index(numLevels - 1)
        - vector<size_t> p_max_row_size(numLevels -1 )
    */
   
    /*
    ##################################################################
    #                           SOLVER                               #
    ##################################################################
    */



    // TODO: remove num_cols
    Solver GMG(d_value, d_index, d_p_value, d_p_index, numLevels, num_rows, max_row_size, p_max_row_size, damp);
    
    GMG.init();
    GMG.set_num_prepostsmooth(1,1);
    GMG.set_convergence_params(1, 1e-99, 1e-10);
    GMG.set_bs_convergence_params(1, 1e-99, 1e-10);
    GMG.set_cycle('V');
    GMG.set_steps(150, 50);
    hipDeviceSynchronize();
    GMG.solve(d_u, d_b);
    // GMG.solve_(d_value, d_index, max_row_size, d_p_value, d_p_index, p_max_row_size, d_u, d_b, numLevels, num_rows);
    // hipDeviceSynchronize();
    
    // GMG.deallocate();    

    /*
    ##################################################################
    #                           TDO                                  #
    ##################################################################
    */

    // TDO algorithm, tdo.cu
    // produces updated d_kai

    // converge?
    double eta = 12.0;
    double beta = 1.0;

    TDO tdo(d_u, d_kai, h, dim, beta, eta, Assembly.getNumElements(), d_A_local);
    tdo.init();
    tdo.innerloop();


}


// print_GPU<<<1,1>>> ( d_res0 );
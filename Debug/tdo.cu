#include "hip/hip_runtime.h"

#include <iostream>
#include "assemble.h"
#include <cmath>
#include "cudakernels.h"
#include "tdo.h"

using namespace std;


TDO::TDO(double* d_u, double* d_kai, double h, size_t dim, double betastar, double etastar, size_t numElements, size_t num_rows, double* d_A_local, vector<size_t*> d_node_index, vector<size_t> N, double rho)
 : m_d_u(d_u), m_d_kai(d_kai), m_h(h), m_dim(dim), m_numElements(numElements), m_num_rows(num_rows), m_d_A_local(d_A_local), m_d_node_index(d_node_index), m_N(N), m_rho(rho), m_etastar(etastar), m_betastar(betastar)
{
    // inner loop frequency, n
    m_n = (6 / m_etastar) * ( m_betastar / (m_h*m_h) );
    m_del_t = 1.0 / m_n;

    

    // // TODO: calculate p_w
    // double g

    // TODO: reduction: calcP_w
    // calcP_w


    // TODO: betastar, etastar
    
    // TODO: del_t = 1.0 if 3D, see paper page 15
    

    // TODO: local volume
    // NOTE: wrong here because you thought m_h here is baselevel's, it's actually the finest level
    // m_local_volume = pow(m_h, m_dim); 


    
}

bool TDO::init()
{
    

    calculateDimensions(m_numElements, m_gridDim, m_blockDim);

        
    CUDA_CALL( hipMalloc( (void**)&m_d_df, sizeof(double) * m_numElements ) );
    CUDA_CALL( hipMemset( m_d_df, 0, sizeof(double) * m_numElements) );

    CUDA_CALL( hipMalloc( (void**)&m_d_uTAu, sizeof(double) * m_num_rows) );
    CUDA_CALL( hipMemset( m_d_uTAu, 0, sizeof(double) * m_num_rows) );

    CUDA_CALL( hipMalloc( (void**)&m_d_temp, sizeof(double) * m_num_rows) );
    CUDA_CALL( hipMemset( m_d_temp, 0, sizeof(double) * m_num_rows) );

    CUDA_CALL( hipMalloc( (void**)&m_d_beta, sizeof(double) ) );
    CUDA_CALL( hipMemset( m_d_beta, 0, sizeof(double)) );

    CUDA_CALL( hipMalloc( (void**)&m_d_eta, sizeof(double) ) );
    CUDA_CALL( hipMemset( m_d_eta, 0, sizeof(double)) );

    CUDA_CALL( hipMalloc( (void**)&m_d_mutex, sizeof(int) ) );

    CUDA_CALL( hipMalloc( (void**)&m_d_lambda_tr, sizeof(double) ) );
    CUDA_CALL( hipMalloc( (void**)&m_d_lambda_l, sizeof(double) ) );
    CUDA_CALL( hipMalloc( (void**)&m_d_lambda_u, sizeof(double) ) );
    CUDA_CALL( hipMalloc( (void**)&m_d_kai_tr, sizeof(double) * m_numElements) );
    CUDA_CALL( hipMalloc( (void**)&m_d_rho_tr, sizeof(double) ) );
    CUDA_CALL( hipMalloc( (void**)&m_d_p_w, sizeof(double) ) );

    CUDA_CALL( hipMemset( m_d_lambda_l, 0, sizeof(double) ) );
    CUDA_CALL( hipMemset( m_d_lambda_tr, 0, sizeof(double) ) );
    CUDA_CALL( hipMemset( m_d_lambda_u, 0, sizeof(double) ) );
    CUDA_CALL( hipMemset( m_d_kai_tr, 0, sizeof(double) * m_numElements) );
    CUDA_CALL( hipMemset( m_d_rho_tr, 0, sizeof(double) ) );
    CUDA_CALL( hipMemset( m_d_p_w, 0, sizeof(double) ) );

    return true;
}

bool TDO::innerloop()
{

    // calculating the driving force of each element
    // df[] = ( 1 / 2*omega ) * ( p * pow(kai[], p - 1 ) ) * sum( u^T * A_local * u * det(J) )

    // TODO: there's no jacobi here, assumed det_J = 1 for now I think
    // CHECK: jacobi maybe is already in A local??
    // CHECK: here temp[] array or scalar?


    // TODO:
        m_local_volume = 0.25*0.25;



    
    // df[] = u^T * A_local * u
    for ( int i = 0 ; i < m_numElements ; i++ )
        calcDrivingForce ( &m_d_df[i], &m_d_kai[i], 3, m_d_uTAu, m_d_u, m_d_node_index[i], m_d_A_local, m_num_rows, m_gridDim, m_blockDim );

    // DEBUG:
        // printVector_GPU<<<1,4>>> ( m_d_A_local, 4 );
        // hipDeviceSynchronize();
        


    // d_temp = u^T * A * u
    vectorEquals_GPU<<<m_gridDim,m_blockDim>>>(m_d_uTAu, m_d_df, m_numElements);

    // NOTE: reduction issue if numElements > blocksize
    calcP_w<<<m_gridDim,m_blockDim>>>(m_d_p_w, m_d_df, m_d_uTAu, m_d_kai, 3, m_local_volume, m_numElements);

    // print_GPU<<<1,1>>>( m_d_p_w );
    // hipDeviceSynchronize();


    // calculate eta and beta
    calcEtaBeta<<<1,2>>>( m_d_eta, m_d_beta, m_etastar, m_betastar, m_d_p_w );
    hipDeviceSynchronize();

    // cout << m_etastar << endl;
    // cout << m_betastar << endl;

    // print_GPU<<<1,1>>>( m_d_eta );
    // hipDeviceSynchronize();
    // print_GPU<<<1,1>>>( m_d_beta );
    // hipDeviceSynchronize();


    // NOTE:
    //// for loop
     for ( int j = 0 ; j < m_n ; j++ )
    {

        // df[] = ( 1 / 2*element_volume ) * p * pow(kai_element, (p-1) ) * temp[]
        // temp[] = u[]^T * A * u[]
        UpdateDrivingForce<<<m_gridDim,m_blockDim>>>( m_d_df, m_d_uTAu, 3, m_d_kai, m_local_volume, m_numElements );

        // printVector_GPU<<<1,m_numElements>>> ( m_d_df, m_numElements );
        // hipDeviceSynchronize();


        // TODO: laplacian_GPU in these kernels only work on sym. matrices
        // esp for north elements, N*N --> Nx * Ny
        // bisection algo: 
        
        setToZero<<<1,1>>>(m_d_lambda_tr, 1);
        calcLambdaLower<<< m_gridDim, m_blockDim >>> (m_d_df, m_d_lambda_l, m_d_mutex, m_d_beta, m_d_kai, m_d_eta, m_N[0], m_numElements);
        calcLambdaUpper<<< m_gridDim, m_blockDim >>> (m_d_df, m_d_lambda_u, m_d_mutex, m_d_beta, m_d_kai, m_d_eta, m_N[0], m_numElements);
        
        
        // print_GPU <<< 1 , 1 >>> ( m_d_eta );
        // print_GPU <<< 1 , 1 >>> ( m_d_beta );
        // hipDeviceSynchronize();



        for ( int i = 0 ; i < 20 ; i++ )
        {
            calcKaiTrial<<<m_gridDim,m_blockDim>>> ( m_d_kai, m_d_df, m_d_lambda_tr, m_del_t, m_d_eta, m_d_beta, m_d_kai_tr, m_N[0], m_numElements);

            // printVector_GPU<<<1,4>>>( m_d_kai_tr, 4);
            // hipDeviceSynchronize();


            setToZero<<<1,1>>>(m_d_rho_tr, 1);
            sumOfVector_GPU <<< m_gridDim, m_blockDim >>> (m_d_rho_tr, m_d_kai_tr, m_numElements);

                    

            // printVector_GPU<<<m_gridDim,m_blockDim>>>( m_d_kai_tr, m_numElements);
            // hipDeviceSynchronize();

            calcLambdaTrial<<<1,1>>>( m_d_rho_tr, m_rho, m_d_lambda_l, m_d_lambda_u, m_d_lambda_tr);
              

        }


        // kai(j) = kai(j+1)
        vectorEquals_GPU<<<m_gridDim,m_blockDim>>>( m_d_kai, m_d_kai_tr, m_numElements );

        


    }

    

    
    return true;

}
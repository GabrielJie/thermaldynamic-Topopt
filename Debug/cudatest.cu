#include "hip/hip_runtime.h"
#include<iostream>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n)
  {
    y[i] = a*x[i] + y[i];       // 16
    
    // for ( int i = 0 ; i < 1 ; i++)
    // {
    //     x[i] = x[i] + 1;       // 8*10
    //     // for ( int j = 0 ; j < 2000 ; j++ )
    //     //     x[i] += 2;
    // }
  } 
      

}

int main(void)
{
  int N = 30 * (1 << 20);
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

    std::cout << "N = " << N << std::endl;

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  hipEventRecord(start);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+511)/512, 512>>>(N, 2.0f, d_x, d_y);

  hipEventRecord(stop);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = max(maxError, abs(y[i]-4.0f));
  }

  printf("Max error: %f\n", maxError);
  printf("Effective Bandwidth (GB/s): %f\n", N*4*(2)/milliseconds/1e6);
  std::cout << "Time taken = " << milliseconds << " ms" << std::endl;
}
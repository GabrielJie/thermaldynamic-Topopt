#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}


// Determines 1-dimensional CUDA block and grid sizes based on the number of rows N
__host__ 
void calculateDimensions(size_t N, dim3 &gridDim, dim3 &blockDim)
{
    if ( N <= 1024 )
    {
        blockDim.x = 1024; blockDim.y = 1; blockDim.z = 1;
        gridDim.x  = 1; gridDim.y = 1; gridDim.z = 1;
    }
        
    else
    {
        blockDim.x = 1024; blockDim.y = 1; blockDim.z = 1;
        gridDim.x  = (int)ceil(N/blockDim.x)+1; gridDim.y = 1; gridDim.z = 1;
    }
}

int main(void)
{
  int N = 20 * (1 << 20);
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

    dim3 gridDim;
    dim3 blockDim;

    calculateDimensions(N, gridDim, blockDim);


  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  hipEventRecord(start);

  // Perform SAXPY on 1M elements
    // saxpy<<<(N+511)/512, 512>>>(N, 2.0f, d_x, d_y);
  saxpy<<<gridDim, blockDim>>>(N, 2.0f, d_x, d_y);

  hipEventRecord(stop);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++) 
  {
    maxError = max(maxError, abs(y[i]-4.0f));
  }

  printf("Time = %f\n", milliseconds);
  printf("Effective Bandwidth (GB/s): %f\n", N*4*3/milliseconds/1e6);

}
#include "hip/hip_runtime.h"

#include <iostream>
#include "assemble.h"
#include <cmath>
#include "cudakernels.h"
#include "tdo.h"

using namespace std;


TDO::TDO(double* d_u, double* d_chi, double h, size_t dim, double betastar, double etastar, size_t numElements, size_t num_rows, double* d_A_local, vector<size_t*> d_node_index, vector<size_t> N, double rho, size_t numLevels, size_t p)
 : m_d_u(d_u), m_d_chi(d_chi), m_h(h), m_dim(dim), m_numElements(numElements), m_num_rows(num_rows), m_d_A_local(d_A_local), m_d_node_index(d_node_index), m_N(N), m_rho(rho), m_etastar(etastar), m_betastar(betastar), m_numLevels(numLevels), m_p(p)
{
    // inner loop frequency, n
    m_n = (6 / m_etastar) * ( m_betastar / (m_h*m_h) );
    m_del_t = 1.0 / m_n;

    
    // // TODO: calculate p_w
    // double g

    // TODO: reduction: calcP_w
    // calcP_w


    // TODO: betastar, etastar
    
    // TODO: del_t = 1.0 if 3D, see paper page 15
    

    // local volume
    // NOTE: wrong here because you thought m_h here is baselevel's, it's actually the finest level
    m_local_volume = pow(m_h, m_dim); 

    
    
}

bool TDO::init()
{

    calculateDimensions(m_numElements, m_gridDim, m_blockDim);

        
    CUDA_CALL( hipMalloc( (void**)&m_d_df, sizeof(double) * m_numElements ) );
    CUDA_CALL( hipMemset( m_d_df, 0, sizeof(double) * m_numElements) );

    CUDA_CALL( hipMalloc( (void**)&m_d_uTAu, sizeof(double) * m_num_rows) );
    CUDA_CALL( hipMemset( m_d_uTAu, 0, sizeof(double) * m_num_rows) );

    CUDA_CALL( hipMalloc( (void**)&m_d_temp, sizeof(double) * m_num_rows) );
    CUDA_CALL( hipMemset( m_d_temp, 0, sizeof(double) * m_num_rows) );

    CUDA_CALL( hipMalloc( (void**)&m_d_beta, sizeof(double) ) );
    CUDA_CALL( hipMemset( m_d_beta, 0, sizeof(double)) );

    CUDA_CALL( hipMalloc( (void**)&m_d_eta, sizeof(double) ) );
    CUDA_CALL( hipMemset( m_d_eta, 0, sizeof(double)) );

    CUDA_CALL( hipMalloc( (void**)&m_d_mutex, sizeof(int) ) );

    CUDA_CALL( hipMalloc( (void**)&m_d_lambda_tr, sizeof(double) ) );
    CUDA_CALL( hipMalloc( (void**)&m_d_lambda_l, sizeof(double) ) );
    CUDA_CALL( hipMalloc( (void**)&m_d_lambda_u, sizeof(double) ) );
    CUDA_CALL( hipMalloc( (void**)&m_d_chi_tr, sizeof(double) * m_numElements) );
    CUDA_CALL( hipMalloc( (void**)&m_d_rho_tr, sizeof(double) ) );
    CUDA_CALL( hipMalloc( (void**)&m_d_p_w, sizeof(double) ) );

    CUDA_CALL( hipMemset( m_d_lambda_l, 0, sizeof(double) ) );
    CUDA_CALL( hipMemset( m_d_lambda_tr, 0, sizeof(double) ) );
    CUDA_CALL( hipMemset( m_d_lambda_u, 0, sizeof(double) ) );
    CUDA_CALL( hipMemset( m_d_chi_tr, 0, sizeof(double) * m_numElements) );
    CUDA_CALL( hipMemset( m_d_rho_tr, 0, sizeof(double) ) );
    CUDA_CALL( hipMemset( m_d_p_w, 0, sizeof(double) ) );

    return true;
}

bool TDO::innerloop(double* &d_u, double* &d_chi)
{
    m_d_u = d_u;
    m_d_chi = d_chi;
    
    // calculating the driving force of each element
    // df[] = ( 1 / 2*omega ) * ( p * pow(chi[], p - 1 ) ) * sum( u^T * A_local * u )
    // df[] = u^T * A_local * u
    for ( int i = 0 ; i < m_numElements ; i++ )
        calcDrivingForce ( &m_d_df[i], &m_d_chi[i], m_p, m_d_uTAu, m_d_u, m_d_node_index[i], m_d_A_local, m_num_rows, m_gridDim, m_blockDim );

    // DEBUG:
        // cout << "m_d_df" << endl;
        // hipDeviceSynchronize();
        // printVector_GPU<<<1,m_numElements>>> ( m_d_df, m_numElements );
        // hipDeviceSynchronize();
        


    // d_temp = u^T * A * u
    vectorEquals_GPU<<<m_gridDim,m_blockDim>>>(m_d_uTAu, m_d_df, m_numElements);

    // NOTE: reduction issue if numElements > blocksize
    calcP_w<<<m_gridDim,m_blockDim>>>(m_d_p_w, m_d_df, m_d_uTAu, m_d_chi, m_p, m_local_volume, m_numElements);

    // print_GPU<<<1,1>>>( m_d_p_w );
    // hipDeviceSynchronize();


    // calculate eta and beta
    calcEtaBeta<<<1,2>>>( m_d_eta, m_d_beta, m_etastar, m_betastar, m_d_p_w );
    hipDeviceSynchronize();

    // cout << m_etastar << endl;
    // cout << m_betastar << endl;

    // print_GPU<<<1,1>>>( m_d_beta );
    // hipDeviceSynchronize();
    // print_GPU<<<1,1>>>( m_d_eta );
    // hipDeviceSynchronize();


    // NOTE:
    //// for loop
     for ( int j = 0 ; j < m_n ; j++ )
    {

        // df[] = ( 1 / 2*element_volume ) * p * pow(chi_element, (p-1) ) * temp[]
        // temp[] = u[]^T * A * u[]
        UpdateDrivingForce<<<m_gridDim,m_blockDim>>>( m_d_df, m_d_uTAu, m_p, m_d_chi, m_local_volume, m_numElements );

        // printVector_GPU<<<1,m_numElements>>> ( m_d_df, m_numElements );
        // hipDeviceSynchronize();


        // TODO: laplacian_GPU in these kernels only work on sym. matrices
        // esp for north elements, N*N --> Nx * Ny
        // bisection algo: 
        
        setToZero<<<1,1>>>(m_d_lambda_tr, 1);
        calcLambdaLower<<< m_gridDim, m_blockDim >>> (m_d_df, m_d_lambda_l, m_d_mutex, m_d_beta, m_d_chi, m_d_eta, m_N[0], m_numElements);
        calcLambdaUpper<<< m_gridDim, m_blockDim >>> (m_d_df, m_d_lambda_u, m_d_mutex, m_d_beta, m_d_chi, m_d_eta, m_N[0], m_numElements);
        
        // hipDeviceSynchronize();
        // cout << "eta, beta" << endl;
        // hipDeviceSynchronize();
        // print_GPU <<< 1 , 1 >>> ( m_d_eta );
        // hipDeviceSynchronize();
        // print_GPU <<< 1 , 1 >>> ( m_d_beta );
        // // printVector_GPU<<<1,m_numElements>>> ( m_d_df, m_numElements );

        for ( int i = 1 ; i < 30 ; i++ )
        {
            // cout << "iteration " << i << endl;
            // hipDeviceSynchronize();
            // cout << "lambda_tr" << endl;
            // print_GPU <<< 1 , 1 >>> ( m_d_lambda_tr );
            // hipDeviceSynchronize();
            // cout << "lambda_l" << endl;
            // print_GPU <<< 1 , 1 >>> ( m_d_lambda_l );
            // hipDeviceSynchronize();
            // cout << "lambda_u" << endl;
            // print_GPU <<< 1 , 1 >>> ( m_d_lambda_u );
            // hipDeviceSynchronize();


            calcChiTrial<<<m_gridDim,m_blockDim>>> ( m_d_chi, m_d_df, m_d_lambda_tr, m_del_t, m_d_eta, m_d_beta, m_d_chi_tr, m_N[0], m_numElements);

            // printVector_GPU<<<1,4>>>( m_d_chi_tr, 4);
            // hipDeviceSynchronize();


            setToZero<<<1,1>>>(m_d_rho_tr, 1);
            sumOfVector_GPU <<< m_gridDim, m_blockDim >>> (m_d_rho_tr, m_d_chi_tr, m_numElements);
            calcRhoTrial<<<1,1>>>(m_d_rho_tr, m_local_volume, m_numElements);

            // // printVector_GPU<<<m_gridDim,m_blockDim>>>( m_d_chi_tr, m_numElements);
            // print_GPU<<<1,1>>>( m_d_rho_tr );
            // hipDeviceSynchronize();
            // cout << "\n";


            calcLambdaTrial<<<1,1>>>( m_d_rho_tr, m_rho, m_d_lambda_l, m_d_lambda_u, m_d_lambda_tr);
              

        }


        // chi(j) = chi(j+1)
        vectorEquals_GPU<<<m_gridDim,m_blockDim>>>( m_d_chi, m_d_chi_tr, m_numElements );
       


    }

    

    
    return true;

}
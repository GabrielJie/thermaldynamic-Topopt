#include <iostream>
#include "assemble.h"
#include <cmath>
#include "cudakernels.h"

using namespace std;


// Node class definitions
Assembler::Node::Node (int id) : m_index(id){}

void Assembler::Node::setXCoor(float x) { m_coo[0] = x; }
void Assembler::Node::setYCoor(float y) { m_coo[1] = y; }
float Assembler::Node::getXCoor(float x) { return m_coo[0]; }
float Assembler::Node::getYCoor(float y) { return m_coo[1]; }
int Assembler::Node::index() { return m_index; }


void Assembler::Node::printCoor()
{
    cout << "node [" << m_index << "] = ( " << m_coo[0] << ", " << m_coo[1] << " )" << endl;
}


// Element class definitions
Assembler::Element::Element(size_t ind) : m_index(ind)
{
    
}

double Assembler::Element::valueAt(size_t x, size_t y, size_t num_cols)
{
    // return m_A_local[ y + x * num_cols ];
    // cout << m_A_local << endl;

    return 0;
}

size_t Assembler::Element::index() { return m_index; }

void Assembler::Element::addNode(Node *x) { m_node.push_back(x); }

int Assembler::Element::nodeIndex(int i) { return m_node[i]->index(); }

void Assembler::Element::printNodes() 
{
    cout << "Element " << m_index << ": nodes = { ";
    for ( int i = 0 ; i < m_node.size() - 1 ; ++i )
        cout << m_node[i]->index() << ", ";

    cout << m_node[ m_node.size() - 1]->index() << " }" << endl;
}



Assembler::Assembler(size_t dim, double youngMod, double poisson)
{
    cout << "assembler" << endl;

    m_youngMod = youngMod;
    m_poisson = poisson;

    m_dim = dim;

}

Assembler::~Assembler()
{
    cout << "assembler : deallocate" << endl;
    // CUDA_CALL( hipFree(d_m_A_local) );
}

bool Assembler::set_domain_size(size_t h, size_t Nx, size_t Ny)
{
    m_h = h;
    m_Nx = Nx;
    m_Ny = Ny;
    return true;
}

bool Assembler::init()
{

    if ( m_dim == 2 )
    {
        m_A_local.resize(64, 0.0);
        m_num_rows_l = 8;
    }

    else if (m_dim == 3 )
    {
        m_A_local.resize(144, 0.0);
        m_num_rows_l = 12;
    }

    else
        cout << "error" << endl; //TODO: add error/assert
    
    // calculate the number of nodes/elements in the domain
    m_numElements = m_Nx * m_Ny;
    
    m_numNodesPerDim.resize(m_dim);

    // TODO: for loop, possibly change Nx, Ny, .. to vector<size_t> numElements
    m_numNodesPerDim[0] = m_Nx + 1;
    m_numNodesPerDim[1] = m_Ny + 1;

    m_numNodes = 1;
    for ( int i = 0 ; i < m_dim ; i++ )
        m_numNodes *= m_numNodesPerDim[i];


    // num of rows in global stiffness matrix
    m_num_rows_g = m_numNodes * m_dim;

    assembleLocal();
    assembleGlobal();
    // cout << m_A_local[0] << endl;

    // CUDA_CALL( hipMalloc((void**)&d_m_A_local, sizeof(double) * m_A_local.size()) );
    // CUDA_CALL( hipMemcpy(d_m_A_local, &m_A_local[0], sizeof(double) * m_A_local.size(), hipMemcpyHostToDevice) );
    
    return true;

}

double Assembler::valueAt(size_t x, size_t y)
{
    return m_A_local[y + x*m_num_rows_l];
}

// to produce an ELLmatrix of the global stiffness in the device
// will return d_value, d_index, d_max_row_size
bool Assembler::assembleGlobal()
{
    // TODO: if no BC is set, return false with error

    for ( int i = 0 ; i < m_numNodes ; ++i )
    {
        m_node.push_back(Node(i));
    }

    for ( int i = 0 ; i < m_numElements ; ++i )
    {
        m_element.push_back(Element(i));
    }

    // assign the nodes to each element
    for ( int i = 0 ; i < m_numElements ; i++ )
    {
        m_element[i].addNode(&m_node[ i + i/m_Nx ]);   // lower left node
        m_element[i].addNode(&m_node[ i + i/m_Nx + 1]);   // lower right node
        m_element[i].addNode(&m_node[ i + i/m_Nx + m_Nx + 1]);   // upper left node
        m_element[i].addNode(&m_node[ i + i/m_Nx + m_Nx + 2]);   // upper right node
    }

    // cout << m_node[0].index() << endl;
    // m_element[0].printNodes();

    // create a function for this so that A_g is temporary
    
    // double A_g[m_numNodes*m_dim][m_numNodes*m_dim];
    vector<vector<double>> A_g (m_numNodes*m_dim, std::vector <double> (m_numNodes*m_dim, 0.0));

    for ( int i = 0 ; i < m_numNodes*m_dim; i++)
    {
        for ( int j = 0 ; j < m_numNodes*m_dim; j++)
            A_g[i][j] = 0;
    }

    cout << m_numElements << endl;
    

    // TODO: !!!
    see TDO sheet 12, check the discrepencies
    think it's due to this for loop below here:


    for ( int elmn_index = 0 ; elmn_index < m_numElements ; elmn_index++ )
    {
        for ( int x = 0 ; x < 4 ; x++ ) // TODO: dim  
        {
            for ( int y = 0 ; y < 4 ; y++ )        // TODO: dim   
            {      

                    A_g[ 2*m_element[elmn_index].nodeIndex(x)     ][ 2*m_element[elmn_index].nodeIndex(y)     ] += valueAt( 2*x    , 2*y );
                    A_g[ 2*m_element[elmn_index].nodeIndex(x)     ][ 2*m_element[elmn_index].nodeIndex(y) + 1 ] += valueAt( 2*x    , 2*y + 1      );
                    A_g[ 2*m_element[elmn_index].nodeIndex(x) + 1 ][ 2*m_element[elmn_index].nodeIndex(y)     ] += valueAt( 2*x + 1, 2*y          );
                    A_g[ 2*m_element[elmn_index].nodeIndex(x) + 1 ][ 2*m_element[elmn_index].nodeIndex(y) + 1 ] += valueAt( 2*x + 1, 2*y + 1      );
            }
        }
    }

    // for ( int i = 0 ; i < m_bc_index.size() ; ++i )
    //     applyMatrixBC(A_g, m_bc_index[i], m_num_rows_g);

    for ( int i = 0 ; i < m_num_rows_g ; i++ )
    {
        for ( int j = 0 ; j < m_num_rows_g ; j++ )
            cout << A_g[i][j] << " ";
        
        cout << "\n";
    }





    // calculate global max_num_rows
    m_max_row_size = getMaxRowSize(A_g, m_num_rows_g);


    // transformtoELL
    // transformToELL(A_g, m_value_g, m_index_g, m_max_row_size, m_num_rows_g);

    // for ( int i = 0 ; i < m_max_row_size * m_num_rows_g ; i++ )
    // cout << m_value_g[i] << " ";

    // cout << "\n";

    





    return true;

}


// TODO: check this is not right!
// assembles the local stiffness matrix
bool Assembler::assembleLocal()
{
    cout << "assembleLocal" << endl;

    // TODO: you haven't added JACOBI, see "TODO:" just before this function's return true

 

    double E[3][3];

    E[0][0] = E[1][1] = m_youngMod/(1 - m_poisson * m_poisson );
    E[0][1] = E[1][0] = m_poisson * E[0][0];
    E[2][2] = (1 - m_poisson) / 2 * E[0][0];
    E[2][0] = E[2][1] = E[1][2] = E[0][2];

    // bilinear shape function matrix (using 4 Gauss Points)
    double B[4][3][8] = { { {-0.3943375,	0,	0.3943375,	0,	0.1056625,	0,	-0.1056625,	0}, {0,	-0.3943375,	0,	-0.1056625,	0,	0.1056625,	0,	0.3943375} , {-0.3943375,	-0.3943375,	-0.1056625,	0.3943375,	0.1056625,	0.1056625,	0.3943375,	-0.1056625} },
                          { {-0.3943375,	0,	0.3943375,	0,	0.1056625,	0,	-0.1056625,	0}, {0,	-0.1056625,	0,	-0.3943375,	0,	0.3943375,	0,	0.1056625}, {-0.1056625,	-0.3943375,	-0.3943375,	0.3943375,	0.3943375,	0.1056625,	0.1056625,	-0.1056625} },
                          { {-0.1056625,	0,	0.1056625,	0,	0.3943375,	0,	-0.3943375,	0}, {0,	-0.3943375,	0,	-0.1056625,	0,	0.1056625,	0,	0.3943375}, {-0.3943375,	-0.1056625,	-0.1056625,	0.1056625,	0.1056625,	0.3943375,	0.3943375,	-0.3943375} },
                          { {-0.1056625,	0,	0.1056625,	0,	0.3943375,	0,	-0.3943375,	0}, {0,	-0.1056625,	0,	-0.3943375,	0,	0.3943375,	0,	0.1056625}, {-0.1056625,	-0.1056625,	-0.3943375,	0.1056625,	0.3943375,	0.3943375,	0.1056625,	-0.3943375} }
                        };

    // 4 matrices with size 3x8 to store each GP's stiffness matrix
    double foo[4][3][8];
    double bar[4][8][8];

    // intializing to zero
    for ( int GP = 0 ; GP < 4 ; GP++)
    {
        for ( int i = 0 ; i < 8 ; i++ )
        {
            for( int j = 0 ; j < 3 ; j++ )
                foo[GP][j][i] = 0;

            for( int j = 0 ; j < 8 ; j++ )
                bar[GP][j][i] = 0;
        }
    }

    // calculating A_local = B^T * E * B

    // foo = E * B
    for ( int GP = 0 ; GP < 4 ; GP++)
    {
        for ( int i = 0 ; i < 3 ; i++ )
        {
            for( int j = 0 ; j < 8 ; j++ )
            {
                for ( int k = 0 ; k < 3 ; k++)
                    foo[GP][i][j] += E[i][k] * B[GP][k][j];
            }
        }
    }

    
    // bar = B^T * foo
    for ( int GP = 0 ; GP < 4 ; GP++)
    {
        for ( int i = 0 ; i < 8 ; i++ )
        {
            for( int j = 0 ; j < 8 ; j++ )
            {
                for ( int k = 0 ; k < 3 ; k++)
                    bar[GP][i][j] += B[GP][k][i] * foo[GP][k][j];
            }
        }
    }


    for ( int GP = 0 ; GP < 4 ; GP++)
    {
        for ( int i = 0 ; i < 8 ; i++ )
        {
            for( int j = 0 ; j < 8 ; j++ )
                m_A_local[j + i*m_num_rows_l] += bar[GP][i][j];     // TODO: * jacobi here
        }
    }


    return true;
}

void Assembler::setBC(vector<size_t> bc_index)
{
    m_bc_index = bc_index;
}

// assembles the local stiffness matrix
vector<double> Assembler::assembleLocal_(double youngMod, double poisson)
{
    cout << "assembleLocal" << endl;
    vector<double> A_local;
    // TODO: you haven't added JACOBI, see "TODO:" just before this function's return true

    size_t num_cols;

    if ( m_dim == 2 )
    {
        A_local.resize(64, 0.0);
        num_cols = 8;
    }

    else if (m_dim == 3 )
    {
        A_local.resize(144, 0.0);
        num_cols = 12;
    }

    else
        cout << "error" << endl; //TODO: add error/assert

    double E[3][3];

    E[0][0] = E[1][1] = youngMod/(1 - poisson * poisson );
    E[0][1] = E[1][0] = poisson * E[0][0];
    E[2][2] = (1 - poisson) / 2 * E[0][0];
    E[2][0] = E[2][1] = E[1][2] = E[0][2];

    // bilinear shape function matrix (using 4 Gauss Points)
    double B[4][3][8] = { { {-0.3943375,	0,	0.3943375,	0,	0.1056625,	0,	-0.1056625,	0}, {0,	-0.3943375,	0,	-0.1056625,	0,	0.1056625,	0,	0.3943375} , {-0.3943375,	-0.3943375,	-0.1056625,	0.3943375,	0.1056625,	0.1056625,	0.3943375,	-0.1056625} },
                          { {-0.3943375,	0,	0.3943375,	0,	0.1056625,	0,	-0.1056625,	0}, {0,	-0.1056625,	0,	-0.3943375,	0,	0.3943375,	0,	0.1056625}, {-0.1056625,	-0.3943375,	-0.3943375,	0.3943375,	0.3943375,	0.1056625,	0.1056625,	-0.1056625} },
                          { {-0.1056625,	0,	0.1056625,	0,	0.3943375,	0,	-0.3943375,	0}, {0,	-0.3943375,	0,	-0.1056625,	0,	0.1056625,	0,	0.3943375}, {-0.3943375,	-0.1056625,	-0.1056625,	0.1056625,	0.1056625,	0.3943375,	0.3943375,	-0.3943375} },
                          { {-0.1056625,	0,	0.1056625,	0,	0.3943375,	0,	-0.3943375,	0}, {0,	-0.1056625,	0,	-0.3943375,	0,	0.3943375,	0,	0.1056625}, {-0.1056625,	-0.1056625,	-0.3943375,	0.1056625,	0.3943375,	0.3943375,	0.1056625,	-0.3943375} }
                        };

    // 4 matrices with size 3x8 to store each GP's stiffness matrix
    double foo[4][3][8];
    double bar[4][8][8];

    // intializing to zero
    for ( int GP = 0 ; GP < 4 ; GP++)
    {
        for ( int i = 0 ; i < 8 ; i++ )
        {
            for( int j = 0 ; j < 3 ; j++ )
                foo[GP][j][i] = 0;

            for( int j = 0 ; j < 8 ; j++ )
                bar[GP][j][i] = 0;
        }
    }

    // calculating A_local = B^T * E * B

    // foo = E * B
    for ( int GP = 0 ; GP < 4 ; GP++)
    {
        for ( int i = 0 ; i < 3 ; i++ )
        {
            for( int j = 0 ; j < 8 ; j++ )
            {
                for ( int k = 0 ; k < 3 ; k++)
                    foo[GP][i][j] += E[i][k] * B[GP][k][j];
            }
        }
    }

    
    // bar = B^T * foo
    for ( int GP = 0 ; GP < 4 ; GP++)
    {
        for ( int i = 0 ; i < 8 ; i++ )
        {
            for( int j = 0 ; j < 8 ; j++ )
            {
                for ( int k = 0 ; k < 3 ; k++)
                    bar[GP][i][j] += B[GP][k][i] * foo[GP][k][j];
            }
        }
    }


    for ( int GP = 0 ; GP < 4 ; GP++)
    {
        for ( int i = 0 ; i < 8 ; i++ )
        {
            for( int j = 0 ; j < 8 ; j++ )
                m_A_local[j + i*num_cols] += bar[GP][i][j];     // TODO: * jacobi here
        }
    }


    return A_local;
}

#include "hip/hip_runtime.h"
/*
    
*/

#include <iostream>
// #include <hip/hip_runtime.h>
#include <vector>
// #include <hip/hip_runtime.h>
// #include "../include/mycudaheader.h"
// #include "precond.h"
#include "cudakernels.h"
#include "assemble.h"
#include "solver.h"
#include "tdo.h"
#include "vtk.h"

using namespace std;

// DONE: bc cases
// DONE: matrix assembly 2D 3D
// DONE: fix prolongation assembly - has something to do with bc initialization
    // DONE: 2D assembly not symmetric

// TODO: store local k matrix in constant memory
// TODO: ApplyTranspose(prol) --> Apply(rest)
// TODO: applyMatrixBC_GPU( valuevector, indexvector, mrs, bcindex(node), "which dimension is free", numrows)
// TODO: __device__ valueAt() has x and y mixed up

// 3D
// TODO: local stiffness
// DONE: 3d elements' node distribution
// TODO: laplacian

//// PARALELLIZABLE
// TODO: fillIndexVector_GPU()

//// LOW PRIORITY
// TODO: VTK class
// TODO: RA and AP's valueAt(indices) are a bit messed up and confusing
// TODO: enum : bc case 
    

int main()
{
  
    // create vtk files
    bool writeToVTK = true;

    // material properties
    double youngMod = 210e9;
    double poisson = 0.3;

    //// model set-up
    size_t numLevels = 2;
    
    vector<size_t> N;
    vector<vector<size_t>> bc_index(numLevels);
    // domain dimensions (x,y,z) on coarsest grid
    N = {3,1,1};

    // local element mesh size on coarsest grid
    double h_coarse = 1;


    size_t dim = N.size();
    bc_index = applyBC(N, numLevels, 0, dim);
    
    // calculating the mesh size on the top level grid
    double h = h_coarse/pow(2,numLevels - 1);

    // smoother (jacobi damping parameter)
    double damp = 2.0/3.0;

    size_t local_num_rows = 4 * dim;

    // TDO
    double rho = 0.3;
    size_t p = 3;
    double etastar = 12.0;
    double betastar = 2 * pow(h,2);

    vector<size_t> num_rows;
    vector<size_t> max_row_size;
    vector<size_t> p_max_row_size;
    vector<size_t> r_max_row_size;

    //// device pointers

    // local stiffness
    double* d_A_local;

    // global stiffness matrix on each grid-level
    vector<double*> d_value;
    vector<size_t*> d_index;

    // prolongation matrices
    vector<double*> d_p_value;
    vector<size_t*> d_p_index;

    // restriction matrices
    vector<double*> d_r_value;
    vector<size_t*> d_r_index;

    // design variable
    double* d_chi;           // NOTE: can alloc this immediately



    //// CUDA
    vector<size_t*> d_node_index;
    // d_node_index.resize(4);

    cout << "### GPU-accelerated Thermodynamic Topology Optimization ###" << endl;
    cout << "Number of Multigrid Levels: " << numLevels << endl;
    cout << "Dimension: " << dim << endl;
    cout << "Coarsest grid size = { " << N[0];
        for ( int i = 1 ; i < dim ; ++i )
            cout << ", " << N[i];
    cout << " }\n";
    cout << "Top-level mesh size = " << h << endl;


    /* ##################################################################
    #                           ASSEMBLY                                #
    ###################################################################*/
       

    Assembler Assembly(dim, h, N, youngMod, poisson, rho, p, numLevels);
    Assembly.setBC(bc_index);
    Assembly.init_GPU(d_A_local, d_value, d_index, d_p_value, d_p_index, d_r_value, d_r_index, d_chi, num_rows, max_row_size, p_max_row_size, r_max_row_size, d_node_index);
    
    // cout << "Top-level number of rows = " << num_rows[numLevels - 1] << endl;
    // cout << "Number of Elements = " << Assembly.getNumElements() << endl;
    // cout << "Assembly ... DONE" << endl;
  
    // // vector u, b
    // vector<double> b(num_rows[numLevels - 1], 0);
    // double force = -1;
    
    // applyLoad(b, N, numLevels, 0, dim, force);



    // double* d_u;
    // double* d_b;
    // // TODO: optimizable: malloc while program is assembling
    // CUDA_CALL( hipMalloc((void**)&d_u, sizeof(double) * num_rows[numLevels - 1] ) );
    // CUDA_CALL( hipMalloc((void**)&d_b, sizeof(double) * num_rows[numLevels - 1] ) );

    // CUDA_CALL( hipMemset(d_u, 0, sizeof(double) * num_rows[numLevels - 1]) );
    // CUDA_CALL( hipMemcpy(d_b, &b[0], sizeof(double) * num_rows[numLevels - 1], hipMemcpyHostToDevice) );




    // /* ##################################################################
    // #                           SOLVER                                  #
    // ###################################################################*/

    // Solver GMG(d_value, d_index, d_p_value, d_p_index, numLevels, num_rows, max_row_size, p_max_row_size, damp);
    
    // // TODO: repair these three, it's a bit messed up
    // GMG.set_convergence_params(100, 1e-99, 1e-15);
    // GMG.set_bs_convergence_params(20, 1e-99, 1e-15);
    // GMG.set_steps(100, 20); 
    

    // GMG.init();
    // GMG.set_verbose(0, 0);
    // GMG.set_num_prepostsmooth(3,3);
    // GMG.set_cycle('V');
    
    // GMG.solve(d_u, d_b, d_value);
    // hipDeviceSynchronize();

    // cout << "Solver   ... DONE" << endl;


    // /* ##################################################################
    // #                           TDO                                     #
    // ###################################################################*/


    // TDO tdo(d_u, d_chi, h, dim, betastar, etastar, Assembly.getNumElements(), local_num_rows, d_A_local, d_node_index, Assembly.getGridSize(), rho, numLevels, p);
    // tdo.init();
    // tdo.set_verbose(0);
    // tdo.innerloop(d_u, d_chi);    // get updated d_chi
    
    // // TODO: create a VTK class, write a function for this to make it neater
    // // vtk stuff
    // vector<double> chi(Assembly.getNumElements(), rho);
    // vector<double> u(Assembly.getNumNodes() * dim, 0);
    // string fileformat(".vtk");
    // int file_index = 0;
    // stringstream ss; 
    // ss << "vtk/tdo";
    // ss << file_index;
    // ss << fileformat;

    // if ( writeToVTK )
    // {
    //     WriteVectorToVTK(chi, u, ss.str(), dim, Assembly.getNumNodesPerDim(), h, Assembly.getNumElements(), Assembly.getNumNodes() );
        
    //     CUDA_CALL( hipMemcpy(&chi[0], d_chi, sizeof(double) * Assembly.getNumElements(), hipMemcpyDeviceToHost) );
    //     CUDA_CALL( hipMemcpy(&u[0], d_u, sizeof(double) * u.size(), hipMemcpyDeviceToHost) );

    //     file_index++;
    //     ss.str( string() );
    //     ss.clear();
    //     ss << "vtk/tdo";
    //     ss << file_index;
    //     ss << fileformat;
        
    //     WriteVectorToVTK(chi, u, ss.str(), dim, Assembly.getNumNodesPerDim(), h, Assembly.getNumElements(), Assembly.getNumNodes() );
    // }

    // for ( int i = 1 ; i < 10 ; ++i )
    // {
    //     // update the global stiffness matrix with the updated density distribution
    //     Assembly.UpdateGlobalStiffness(d_chi, d_value, d_index, d_p_value, d_p_index, d_r_value, d_r_index, d_A_local);


    //     // TODO: something's wrong with the solver for N = {3,1}
    //     cout << "Calculating iteration " << i << " ... ";
    //     hipDeviceSynchronize();
    //     GMG.reinit();
    //     GMG.set_verbose(0, 0);
    //     // GMG.set_convergence_params(5, 1e-99, 1e-10); // DEBUG:
    //     // GMG.set_steps(5, 2);
    //     GMG.solve(d_u, d_b, d_value);
    //     hipDeviceSynchronize();

    //     // printVector_GPU<<<1,num_rows[numLevels - 1]>>>( d_u, num_rows[numLevels - 1]);
    //     // print_GPU<<<1,1>>>( &d_u[128]);
    //     hipDeviceSynchronize();
    //     // if (result)


    //     // tdo.set_verbose(1);
    //     tdo.innerloop(d_u, d_chi);
        
    //     // hipDeviceSynchronize();
    //     // printVector_GPU<<<1,Assembly.getNumElements()>>>( d_chi, Assembly.getNumElements());
    //     // cout << "\n";

    //     if ( writeToVTK )
    //     { 
    //         CUDA_CALL( hipMemcpy(&chi[0], d_chi, sizeof(double) * Assembly.getNumElements(), hipMemcpyDeviceToHost) );
    //         CUDA_CALL( hipMemcpy(&u[0], d_u, sizeof(double) * u.size(), hipMemcpyDeviceToHost) );

    //         file_index++;
    //         ss.str( string() );
    //         ss.clear();
    //         ss << "vtk/tdo";
    //         ss << file_index;
    //         ss << fileformat;
            
    //         WriteVectorToVTK(chi, u, ss.str(), dim, Assembly.getNumNodesPerDim(), h, Assembly.getNumElements(), Assembly.getNumNodes() );

    //     }
    //     cout << "SUCCESS\n";
    //     hipDeviceSynchronize();
    // }

    hipDeviceSynchronize();
}

    // PTAP_GPU consider using 2d blocks? :
    // https://www.quantstart.com/articles/Matrix-Matrix-Multiplication-on-the-GPU-with-Nvidia-CUDA/

// print_GPU<<<1,1>>> ( d_res0 );
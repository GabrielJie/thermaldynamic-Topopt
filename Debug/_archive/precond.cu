#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

// #include "../include/mycudaheader.h"
// #include "precond.h"

using namespace std;


__global__ 
void Jacobi_Precond_GPU(double* c, double* value, double* r, size_t num_rows)
{

	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if ( id < num_rows )
		c[id] = value[id] * r[id];
}
#include "hip/hip_runtime.h"
/*
    
*/

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime.h>
#include "../include/mycudaheader.h"
#include <cmath>

using namespace std;


__global__
void calcInnerLoop(double* n, double h, double* eta, double* beta)
{
    *n = ( 6 / *eta ) * ( *beta / ( h * h ) );
}


int main()
{
    
    double eta = 12;
    double beta = 1.5;
    double h = 0.5;

    // CUDA
    double *d_eta;
    double *d_n;
    double *d_beta;

    hipMalloc( (void**)&d_eta, sizeof(double) );
    hipMalloc( (void**)&d_n, sizeof(double) );
    hipMalloc( (void**)&d_beta, sizeof(double) );
    
    hipMemset( d_n, 0, sizeof(double) );
    
    hipMemcpy(d_eta, &eta, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, &beta, sizeof(double), hipMemcpyHostToDevice);
    
    // kernel
    calcInnerLoop<<<1,1>>>( d_n, h, d_eta, d_beta );


    
    
    print_GPU<<<1,1>>>(d_n);
    hipDeviceSynchronize();
    
}
#include <iostream>
#include "assemble.h"
#include <cmath>
#include "cudakernels.h"

using namespace std;


// Node class definitions
Assembler::Node::Node (int id) : m_index(id){}

void Assembler::Node::setXCoor(float x) { m_coo[0] = x; }
void Assembler::Node::setYCoor(float y) { m_coo[1] = y; }
float Assembler::Node::getXCoor(float x) { return m_coo[0]; }
float Assembler::Node::getYCoor(float y) { return m_coo[1]; }
int Assembler::Node::index() { return m_index; }


void Assembler::Node::printCoor()
{
    cout << "node [" << m_index << "] = ( " << m_coo[0] << ", " << m_coo[1] << " )" << endl;
}


// Element class definitions
Assembler::Element::Element(int ind) : m_index(ind){}

double Assembler::Element::valueAt(size_t x, size_t y, size_t num_cols)
{
    // return m_A_local[ y + x * num_cols ];
    // cout << m_A_local << endl;

    return 0;
}

size_t Assembler::Element::index() { return m_index; }

void Assembler::Element::addNode(Node *x) { m_node.push_back(x); }

int Assembler::Element::nodeIndex(int i) { return m_node[i]->index(); }

void Assembler::Element::printNodes() 
{
    cout << "Element " << m_index << ": nodes = { ";
    for ( int i = 0 ; i < m_node.size() - 1 ; ++i )
        cout << m_node[i]->index() << ", ";

    cout << m_node[ m_node.size() - 1]->index() << " }" << endl;
}



Assembler::Assembler(size_t dim, size_t h, vector<size_t> N, double youngMod, double poisson, size_t numLevels)
    : m_dim(dim), m_h(h), m_youngMod(youngMod), m_poisson(poisson), m_numLevels(numLevels)
{
    cout << "assembler" << endl;

    

    // m_N [lev][dim]
    // e.g., m_N[lev=1][dim=0] = number of elements in x-dimension on grid-level 1
    m_N.resize(m_numLevels, vector<size_t>(m_dim));

    // storing the grid dimensions of each level
    for ( int lev = 0 ; lev < m_numLevels ; lev++ )
    {
        for ( int i = 0 ; i < m_dim ; i++ )
        {
            m_N[lev][i] = N[i];
            N[i] *= 2;
        }
    }

  


}

Assembler::~Assembler()
{
    cout << "assembler : deallocate" << endl;
    // CUDA_CALL( hipFree(d_m_A_local) );
}


void Assembler::setBC(vector<size_t> bc_index)
{
    m_bc_index = bc_index;
}

bool Assembler::init()
{


    if ( m_dim == 2 )
    {
        m_A_local.resize(64, 0.0);
        m_num_rows_l = 8;
    }

    else if (m_dim == 3 )
    {
        m_A_local.resize(144, 0.0);
        m_num_rows_l = 12;
    }

    else
        cout << "error" << endl; //TODO: add error/assert


    m_topLev = m_numLevels - 1;


    // TODO: perhaps combine these for loops into one? would it work?    
    // number of elements in each grid-level
    m_numElements.resize(m_numLevels, 1);
    
    for ( int lev = 0 ; lev < m_numLevels ; lev++ )
    {
        for ( int i = 0 ; i < m_dim ; i++)
            m_numElements[lev] *= m_N[lev][i];
    }


    // m_numNodesPerDim[lev][dim]
    m_numNodesPerDim.resize(m_numLevels, vector<size_t>(m_dim));

    for ( int lev = 0 ; lev < m_numLevels ; lev++ )
    {
        for ( int i = 0 ; i < m_dim ; i++)
            m_numNodesPerDim[lev][i] = m_N[lev][i] + 1;
    }

    // number of nodes per grid-level
    m_numNodes.resize(m_numLevels, 1);
    // m_numNodes = 1;

    for ( int lev = 0 ; lev < m_numLevels ; lev++ )
    {
        for ( int i = 0 ; i < m_dim ; i++ )
            m_numNodes[lev] *= m_numNodesPerDim[lev][i];
    }


    // num of rows in global stiffness matrix
    m_num_rows.resize(m_numLevels);

    for ( int lev = 0 ; lev < m_numLevels ; lev++ )
        m_num_rows[lev] = m_numNodes[lev] * m_dim;


    assembleLocal();

    assembleGlobal();

    // // CUDA_CALL( hipMalloc((void**)&d_m_A_local, sizeof(double) * m_A_local.size()) );
    // // CUDA_CALL( hipMemcpy(d_m_A_local, &m_A_local[0], sizeof(double) * m_A_local.size(), hipMemcpyHostToDevice) );
    
    return true;

}


// TODO: check this is not right!
// assembles the local stiffness matrix
bool Assembler::assembleLocal()
{
    cout << "assembleLocal" << endl;

    // TODO: you haven't added JACOBI, see "TODO:" just before this function's return true


    double E[3][3];

    E[0][0] = E[1][1] = m_youngMod/(1 - m_poisson * m_poisson );
    E[0][1] = E[1][0] = m_poisson * E[0][0];
    E[2][2] = (1 - m_poisson) / 2 * E[0][0];
    E[2][0] = E[2][1] = E[1][2] = E[0][2];

    // bilinear shape function matrix (using 4 Gauss Points)
    double B[4][3][8] = { { {-0.3943375,	0,	0.3943375,	0,	0.1056625,	0,	-0.1056625,	0}, {0,	-0.3943375,	0,	-0.1056625,	0,	0.1056625,	0,	0.3943375} , {-0.3943375,	-0.3943375,	-0.1056625,	0.3943375,	0.1056625,	0.1056625,	0.3943375,	-0.1056625} },
                          { {-0.3943375,	0,	0.3943375,	0,	0.1056625,	0,	-0.1056625,	0}, {0,	-0.1056625,	0,	-0.3943375,	0,	0.3943375,	0,	0.1056625}, {-0.1056625,	-0.3943375,	-0.3943375,	0.3943375,	0.3943375,	0.1056625,	0.1056625,	-0.1056625} },
                          { {-0.1056625,	0,	0.1056625,	0,	0.3943375,	0,	-0.3943375,	0}, {0,	-0.3943375,	0,	-0.1056625,	0,	0.1056625,	0,	0.3943375}, {-0.3943375,	-0.1056625,	-0.1056625,	0.1056625,	0.1056625,	0.3943375,	0.3943375,	-0.3943375} },
                          { {-0.1056625,	0,	0.1056625,	0,	0.3943375,	0,	-0.3943375,	0}, {0,	-0.1056625,	0,	-0.3943375,	0,	0.3943375,	0,	0.1056625}, {-0.1056625,	-0.1056625,	-0.3943375,	0.1056625,	0.3943375,	0.3943375,	0.1056625,	-0.3943375} }
                        };

    // 4 matrices with size 3x8 to store each GP's stiffness matrix
    double foo[4][3][8];

    // TODO: use std::vector!!!
    // intializing to zero
    for ( int GP = 0 ; GP < 4 ; GP++)
    {
        for ( int i = 0 ; i < 8 ; i++ )
        {
            for( int j = 0 ; j < 3 ; j++ )
                foo[GP][j][i] = 0;

        }
    }

    /// calculating A_local = B^T * E * B
    // foo = E * B
    for ( int GP = 0 ; GP < 4 ; GP++)
    {
        for ( int i = 0 ; i < 3 ; i++ )
        {
            for( int j = 0 ; j < 8 ; j++ )
            {
                for ( int k = 0 ; k < 3 ; k++)
                    foo[GP][i][j] += E[i][k] * B[GP][k][j];
            }
        }
    }

    // bar = B^T * foo
    for ( int GP = 0 ; GP < 4 ; GP++)
    {
        for ( int i = 0 ; i < 8 ; i++ )
        {
            for( int j = 0 ; j < 8 ; j++ )
            {
                for ( int k = 0 ; k < 3 ; k++)
                    m_A_local[j + i*m_num_rows_l] += B[GP][k][i] * foo[GP][k][j];
            }
        }
    }
    
    return true;
}


double Assembler::valueAt(size_t x, size_t y)
{
    return m_A_local[y + x*m_num_rows_l];
}

// to produce an ELLmatrix of the global stiffness in the device
// will return d_value, d_index, d_max_row_size
bool Assembler::assembleGlobal()
{
    // TODO: if no BC is set, return false with error

    // adding nodes and elements to the top-level global grid
    for ( int i = 0 ; i < m_numNodes[m_topLev] ; ++i )
        m_node.push_back(Node(i));

    for ( int i = 0 ; i < m_numElements[m_topLev] ; ++i )
        m_element.push_back(Element(i));


    // assign the nodes to each element
    for ( int i = 0 ; i < m_numElements[m_topLev] ; i++ )
    {
        m_element[i].addNode(&m_node[ i + i/m_N[m_topLev][0] ]);   // lower left node
        m_element[i].addNode(&m_node[ i + i/m_N[m_topLev][0] + 1]);   // lower right node
        m_element[i].addNode(&m_node[ i + i/m_N[m_topLev][0] + m_N[m_topLev][0] + 1]);   // upper left node
        m_element[i].addNode(&m_node[ i + i/m_N[m_topLev][0] + m_N[m_topLev][0] + 2]);   // upper right node
    }

    
    // // double A_g[m_numNodes*m_dim][m_numNodes*m_dim];
    // // TODO: figure out if you keep this as member var or not
    // m_A_g.resize(m_numNodes[m_topLev]*m_dim, vector<double>(m_numNodes[m_topLev]*m_dim));

    // resizing the global stiffness matrices on each grid-level
    m_A_g.resize(m_numLevels);

    for ( int lev = 0 ; lev < m_numLevels ; lev++ )
    {
        // number of columns in each level
        m_A_g[lev].resize(m_num_rows[lev]);
        
        // number of rows in each level
        for ( int j = 0 ; j < m_num_rows[lev] ; j++ )
                m_A_g[lev][j].resize(m_num_rows[lev]);
    }

    // filling in the global stiffness matrix from the local stiffness matrices of the 4 Gauss-Points
    for ( int elmn_index = 0 ; elmn_index < 4 ; elmn_index++ )
    {
        for ( int x = 0 ; x < 4 ; x++ ) // TODO: dim  
        {
            for ( int y = 0 ; y < 4 ; y++ )        // TODO: dim   
            {      

                    m_A_g[m_topLev][ 2*m_element[elmn_index].nodeIndex(x)     ][ 2*m_element[elmn_index].nodeIndex(y)     ] += valueAt( 2*x    , 2*y );
                    m_A_g[m_topLev][ 2*m_element[elmn_index].nodeIndex(x)     ][ 2*m_element[elmn_index].nodeIndex(y) + 1 ] += valueAt( 2*x    , 2*y + 1      );
                    m_A_g[m_topLev][ 2*m_element[elmn_index].nodeIndex(x) + 1 ][ 2*m_element[elmn_index].nodeIndex(y)     ] += valueAt( 2*x + 1, 2*y          );
                    m_A_g[m_topLev][ 2*m_element[elmn_index].nodeIndex(x) + 1 ][ 2*m_element[elmn_index].nodeIndex(y) + 1 ] += valueAt( 2*x + 1, 2*y + 1      );
            }
        }
    }


    
    // TODO: make a function for this
    // replacing any values <1e-7 to 0.0
    for ( int x = 0 ; x < m_numNodes[m_topLev]*m_dim ; x++ ) // TODO: dim  
    {
        for ( int y = 0 ; y < m_numNodes[m_topLev]*m_dim ; y++ )        // TODO: dim   
        {      
            if ( m_A_g[m_topLev][x][y] < 1e-7 && m_A_g[m_topLev][x][y] > -1e-7)
                m_A_g[m_topLev][x][y] = 0.0;
        }
    }

    //     // cout << m_A_g[15][14] << endl;

    // applying BC on the matrix
    // DOFs which are affected by BC will have identity rows { 0 0 .. 1 .. 0 0}
    for ( int i = 0 ; i < m_bc_index.size() ; ++i )
        applyMatrixBC(m_A_g[m_topLev], m_bc_index[i], m_num_rows[m_topLev]);


    /// resizing the prolongation matrices according to the number of grid-levels
    m_P.resize( m_numLevels - 1 );

    // CHECK: not sure!
    for ( int lev = 0 ; lev < m_numLevels - 1 ; lev++)
    {
        // number of columns in each level
        m_P[lev].resize(m_num_rows[lev]);
        
        // number of rows in each level
        for ( int j = 0 ; j < m_num_rows[lev] ; j++ )
                m_P[lev][j].resize(m_num_rows[lev + 1]);
    }


    // TODO: create a function to assemble the prolongation matrices in each level
    // assembleProlMatrices( m_P, m_numLevels )


    // DEBUG: temporary prolong
    m_P[0] =   {
            {1,	0,	0,	0,	0,	0,	0,	0},
            {0,	1,	0,	0,	0,	0,	0,	0},
            {0,	0,	0.5,	0,	0,	0,	0,	0},
            {0,	0,	0,	0.5,	0,	0,	0,	0},
            {0,	0,	1,	0,	0,	0,	0,	0},
            {0,	0,	0,	1,	0,	0,	0,	0},
            {0,	0,	0,	0,	0,	0,	0,	0},
            {0,	0,	0,	0,	0,	0,	0,	0},
            {0,	0,	0.25,	0,	0,	0,	0.25,	0},
            {0,	0,	0,	0.25,	0,	0,	0,	0.25},
            {0,	0,	0.5,	0,	0,	0,	0.5,	0},
            {0,	0,	0,	0.5,	0,	0,	0,	0.5},
            {0,	0,	0,	0,	1,	0,	0,	0},
            {0,	0,	0,	0,	0,	1,	0,	0},
            {0,	0,	0,	0,	0,	0,	0.5,	0},
            {0,	0,	0,	0,	0,	0,	0,	0.5},
            {0,	0,	0,	0,	0,	0,	1,	0},
            {0,	0,	0,	0,	0,	0,	0,	1}
            };


    // for ( int i = 0 ; i < m_num_rows[1] ; i++ )
    // {
    //     for ( int j = 0 ; j < m_num_rows[0] ; j++ )
    //         cout << m_P[0][i][j] << " ";

    //     cout << "\n";
    // }


    // TODO: transform later, after you get the A's in all levels
    



    // for ( int i = 0 ; i < m_num_rows[1] ; i++ )
    // {
    //     for ( int j = 0 ; j < m_num_rows[1] ; j++ )
    //         cout << m_A_g[1][i][j] << " ";

    //     cout << "\n";
    // }

    // // DEBUG: temp
    // std::vector<std::vector<double>> A_coarse ( m_num_rows_l, std::vector <double> (m_num_rows_g, 0.0));

    // // for ( int i = 0 ; i < 4 ; i++ )
    // // {
    // //     for ( int j = 0 ; j < 4 ; j++ )
    // //         cout << A_coarse[i][j] << " ";

    // //     cout << "\n";
    // // }

    // resizing the coarse stiffness matrices on each grid-level

    for ( int lev = 0 ; lev < m_numLevels - 1; lev++ )
        PTAP(m_A_g[lev], m_A_g[lev+1], m_P[lev], m_num_rows[lev+1], m_num_rows[lev] );

    // for ( int i = 0 ; i < m_num_rows[0] ; i++ )
    // {
    //     for ( int j = 0 ; j < m_num_rows[0] ; j++ )
    //         cout << m_A_g[0][i][j] << " ";

    //     cout << "\n";
    // }


    // TODO:
    // create global element ... Element global(-1) : if ind = -1, this is global
    // in this element, store all the node indices
    // have a function to find the N S E W nodes
    
    // Node getNeighbourNode(Node node, enum NSEW)

    // size_t node_index = getNeighbourNode()

    // A[ test.index ][ ] or something like that

    // use this to get the Node, and then 



    // // TODO: maybe do this together with the lower levels?
    // // calculate global max_num_rows, which will also be needed when allocating memory in device
    m_max_row_size.resize(m_numLevels);
    for ( int lev = 0 ; lev < m_numLevels ; lev++ )
        m_max_row_size[lev] = getMaxRowSize(m_A_g[lev], m_num_rows[lev], m_num_rows[lev]);

    m_p_max_row_size.resize ( m_numLevels - 1 );
    for ( int lev = 0 ; lev < m_numLevels - 1 ; lev++ )
        m_p_max_row_size[lev] = getMaxRowSize(m_P[lev], m_num_rows[lev+1], m_num_rows[lev]);
    
    cout << m_max_row_size[0] << endl;
    cout << m_p_max_row_size[0] << endl;
    // obtaining the ELLPACK value and index vectors from the global stiffness matrix
    
    m_p_value_g.resize( m_numLevels - 1 );
    m_p_index_g.resize( m_numLevels - 1 );
    // prolongation matrices
    for ( int lev = 0 ; lev < m_numLevels - 1 ; lev++ )
    {
        transformToELL(m_P[lev], m_p_value_g[lev], m_p_index_g[lev], m_p_max_row_size[lev], m_num_rows[lev+1]);    
    }


    // transformToELL(m_A_g, m_value_g, m_index_g, m_max_row_size, m_num_rows_g);

    for ( int i = 0 ; i < 36 ; i++ )
    {
        cout << m_p_value_g[0][i] << " ";
    }
        cout << "\n";

    // for ( int i = 0 ; i < m_num_rows[1] ; i++ )
    // {
    //     for ( int j = 0 ; j < m_num_rows[0] ; j++ )
    //         cout << m_P[0][i][j] << " ";

    //     cout << "\n";
    // }
    


    // NOTE: can somehow do init for solving now while allocating memory in device?
    // do async malloc then your init() should be AFTER the memcpy stuff, not before


    // CUDA
    // allocating memory in device

    // // local stiffness
    // CUDA_CALL( hipMalloc((void**)&d_m_A_local, sizeof(double) * m_A_local.size() ) );

    // // global matrices on each grid-level
    // for ( int lev = 0 ; lev < m_numLevels ; lev++ )
    // {
    //     CUDA_CALL( hipMalloc((void**)&m_value_g[lev], sizeof(double) * m_max_row_size[lev] * m_num_rows[lev] ) );
    //     CUDA_CALL( hipMalloc((void**)&m_index_g[lev], sizeof(size_t) * m_max_row_size[lev] * m_num_rows[lev] ) );
    // }


    return true;

}



// // assembles the local stiffness matrix
// vector<double> Assembler::assembleLocal_(double youngMod, double poisson)
// {
    // cout << "assembleLocal" << endl;
    // vector<double> A_local;
    // // TODO: you haven't added JACOBI, see "TODO:" just before this function's return true

    // size_t num_cols;

    // if ( m_dim == 2 )
    // {
    //     A_local.resize(64, 0.0);
    //     num_cols = 8;
    // }

    // else if (m_dim == 3 )
    // {
    //     A_local.resize(144, 0.0);
    //     num_cols = 12;
    // }

    // else
    //     cout << "error" << endl; //TODO: add error/assert

    // double E[3][3];

    // E[0][0] = E[1][1] = youngMod/(1 - poisson * poisson );
    // E[0][1] = E[1][0] = poisson * E[0][0];
    // E[2][2] = (1 - poisson) / 2 * E[0][0];
    // E[2][0] = E[2][1] = E[1][2] = E[0][2];

    // // bilinear shape function matrix (using 4 Gauss Points)
    // double B[4][3][8] = { { {-0.3943375,	0,	0.3943375,	0,	0.1056625,	0,	-0.1056625,	0}, {0,	-0.3943375,	0,	-0.1056625,	0,	0.1056625,	0,	0.3943375} , {-0.3943375,	-0.3943375,	-0.1056625,	0.3943375,	0.1056625,	0.1056625,	0.3943375,	-0.1056625} },
    //                       { {-0.3943375,	0,	0.3943375,	0,	0.1056625,	0,	-0.1056625,	0}, {0,	-0.1056625,	0,	-0.3943375,	0,	0.3943375,	0,	0.1056625}, {-0.1056625,	-0.3943375,	-0.3943375,	0.3943375,	0.3943375,	0.1056625,	0.1056625,	-0.1056625} },
    //                       { {-0.1056625,	0,	0.1056625,	0,	0.3943375,	0,	-0.3943375,	0}, {0,	-0.3943375,	0,	-0.1056625,	0,	0.1056625,	0,	0.3943375}, {-0.3943375,	-0.1056625,	-0.1056625,	0.1056625,	0.1056625,	0.3943375,	0.3943375,	-0.3943375} },
    //                       { {-0.1056625,	0,	0.1056625,	0,	0.3943375,	0,	-0.3943375,	0}, {0,	-0.1056625,	0,	-0.3943375,	0,	0.3943375,	0,	0.1056625}, {-0.1056625,	-0.1056625,	-0.3943375,	0.1056625,	0.3943375,	0.3943375,	0.1056625,	-0.3943375} }
    //                     };

    // // 4 matrices with size 3x8 to store each GP's stiffness matrix
    // double foo[4][3][8];
    // double bar[4][8][8];

    // // intializing to zero
    // for ( int GP = 0 ; GP < 4 ; GP++)
    // {
    //     for ( int i = 0 ; i < 8 ; i++ )
    //     {
    //         for( int j = 0 ; j < 3 ; j++ )
    //             foo[GP][j][i] = 0;

    //         for( int j = 0 ; j < 8 ; j++ )
    //             bar[GP][j][i] = 0;
    //     }
    // }

    // // calculating A_local = B^T * E * B

    // // foo = E * B
    // for ( int GP = 0 ; GP < 4 ; GP++)
    // {
    //     for ( int i = 0 ; i < 3 ; i++ )
    //     {
    //         for( int j = 0 ; j < 8 ; j++ )
    //         {
    //             for ( int k = 0 ; k < 3 ; k++)
    //                 foo[GP][i][j] += E[i][k] * B[GP][k][j];
    //         }
    //     }
    // }

    
    // // bar = B^T * foo
    // for ( int GP = 0 ; GP < 4 ; GP++)
    // {
    //     for ( int i = 0 ; i < 8 ; i++ )
    //     {
    //         for( int j = 0 ; j < 8 ; j++ )
    //         {
    //             for ( int k = 0 ; k < 3 ; k++)
    //                 bar[GP][i][j] += B[GP][k][i] * foo[GP][k][j];
    //         }
    //     }
    // }


    // for ( int GP = 0 ; GP < 4 ; GP++)
    // {
    //     for ( int i = 0 ; i < 8 ; i++ )
    //     {
    //         for( int j = 0 ; j < 8 ; j++ )
    //             m_A_local[j + i*num_cols] += bar[GP][i][j];     // TODO: * jacobi here
    //     }
    // }


//     return A_local;
// }

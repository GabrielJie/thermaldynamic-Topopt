#include "hip/hip_runtime.h"
/*
    
*/


#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime.h>
#include "../include/mycudaheader.h"

using namespace std;


__global__ 
void getMaxRowSize(double *array, size_t *max, int *mutex, size_t n)
{
	unsigned int id = threadIdx.x + blockIdx.x*blockDim.x; // 0 - 24
	unsigned int stride = gridDim.x*blockDim.x; // 5
	// unsigned int offset = 0;

    __shared__ size_t local_nnz[1024];

    local_nnz[id] = 0;

    // CHECK: something's fishy here
    // get nnz of each row
        for ( int j = 0 ; j < stride ; j++)
        {
            if ( array[j + stride*id] != 0)
            local_nnz[id]++;
        }

	// reduction
	unsigned int i = blockDim.x/2;
    while(i != 0)
    {
		if(threadIdx.x < i){
			local_nnz[threadIdx.x] = fmaxf(local_nnz[threadIdx.x], local_nnz[threadIdx.x + i]);
		}
		__syncthreads();
		i /= 2;
	}

    if(threadIdx.x == 0)
    {
		while(atomicCAS(mutex,0,1) != 0);  //lock
		*max = fmaxf(*max, local_nnz[0]);
        atomicExch(mutex, 0);  //unlock
    }

}





__global__
void convertToELL(double *coo, double *ell_val, size_t *ell_ind, size_t max_row_size, size_t N)
{
	unsigned int id = threadIdx.x + blockIdx.x*blockDim.x; // 0 - 5
    unsigned int stride = gridDim.x*blockDim.x; // 6
    
    extern __shared__ size_t temp[];
    
        
        // Inputting the values into ELLPACK index vector

        
        // fill in temp vector with values of COO's each row

        for ( int i = 0 ; i < N ; ++i )
        {
            if(id == 0)
                printf("id*stride + i = %d\n", id*stride + i);

            if ( coo[ id*stride + i ] == 0 )    // TODO: replace 0.0 with something else
                temp[ id*stride + i ] = N;

            else
                temp[ id*stride + i ] = i;
        }
       
        int ind_counter = 0;
        for ( int i = 0 ; i < N || ind_counter == max_row_size ; i++ )
        {
            if ( temp[ id*stride + i ] != N )
            {
                temp[ id*stride + ind_counter ] = temp[ id*stride + i ];
                // temp[ id*stride + i ] = N;
                ind_counter++;
            }
            
            else{}
        }
        

             
    size_t counter = 0;

    // NOTE: potentially, include arranging the temp[] or ell_ind() here as well?
    // Compressing the COO, so that the NNZ are located in the first columns
    for ( int i = 0 ; i < stride ; i++ )
    {
        if ( coo[ i + stride*id ] != 0 )    // TODO: change from 0.0 to abs(value) > 0
        {
            coo[counter + stride*id] = coo[ i + stride*id ];

            if ( counter + stride*id != i + stride*id )
                coo[ i + stride*id ] = 0;

            // coo[ i + stride*id ] = 0;
            counter++;
        }
        
        else{}
    }

    // Inputting the values into ELLPACK value vector
    if ( id < max_row_size )
    {
        for ( int i = 0 ; i < N ; ++i )
        {
            ell_val[ id*stride + i ] = coo [ id + stride*i ];
            ell_ind[ id*stride + i ] = temp [ id + stride*i ];
        }
    }



}


class Node
{
public:
    Node (int id) : m_index(id){}


    void setXCoor(float x) { m_coo[0] = x;}
    void setYCoor(float y) { m_coo[1] = y;}
    float getXCoor(float x) { return m_coo[0];}
    float getYCoor(float y) { return m_coo[1];}

    void printCoor()
    {
        cout << "node [" << m_index << "] = ( " << m_coo[0] << ", " << m_coo[1] << " )" << endl;
    }
    int index() 
    { 
        return m_index; 
    }

   

private:
    int m_index;
    float m_coo[2];
    int m_dof[2];
    // vector<int> m_dof(2);
};



class Element
{
public:
    
    // global 
    Element()
    {
        m_vValue.resize(72);
        m_vIndex.resize(72);

        m_max_row_size = 4;
        m_num_rows = 18;
    }
    
    // local element
    Element(size_t ind) : m_index(ind)  // TODO: change int -> size_t
    {   

            m_vValue.resize(25);
            m_vIndex.resize(25);
            
            m_vValue = {4, 	1, 	0, 	1, 	4, 	1, 	1, 	4, 	1, 	1, 	4, 	1, 	1, 	4, 	1, 	1, 	4, 	1, 	1, 	4, 	1, 	1, 	4, 	0};
            m_vIndex = {0, 	1, 	8, 	0, 	1, 	2, 	1, 	2, 	3, 	2, 	3, 	4, 	3, 	4, 	5, 	4, 	5, 	6, 	5, 	6, 	7, 	6, 	7, 	8};

            m_max_row_size = 3;
            m_num_rows = 4;

    }
    

    size_t index()
    {
        return m_index;
    }

    void addNode(Node *x)
    {
        m_node.push_back(x);
        m_node_index_list.push_back(x->index());
    }

    void printNodes()
    {
        cout << "Element " << m_index << endl;
        for ( int i = 0 ; i < m_node.size() ; ++i )
            m_node[i]->printCoor();
    }

    double* getValueAddress() { return &m_vValue[0]; }
    size_t* getIndexAddress() { return &m_vIndex[0]; }

    size_t* getNodeGlobalIndex() { return &m_node_index_list[0]; }

    size_t max_row_size() { return m_max_row_size; }
    size_t num_rows() { return m_num_rows; }

    int nodeIndex(int i)
    {
        return m_node[i]->index();
    }

    double operator()(size_t x, size_t j) 
    {
        return m_K[x][j];
    }



private:
    std::vector<Node*> m_node;
    size_t m_index;
    size_t m_max_row_size;
    size_t m_num_rows;
    vector<size_t> m_node_index_list;
    double m_rho;

    double m_K[8][8];   // TODO: change 8 to dimension-friendly variable
    vector<double> m_vValue;
    vector<size_t> m_vIndex;


};

// returns value at A(x,y)
__device__
double valueAt(size_t x, size_t y, double* vValue, size_t* vIndex, size_t max_row_size)
{
    for(size_t k = 0; k < max_row_size; ++k)
    {
        if(vIndex[x * max_row_size + k] == y)
            return vValue[x * max_row_size + k];
    }

    return 0.0;
}

// A(x,y) = value
__device__
void setAt( size_t x, size_t y, double* vValue, size_t* vIndex, size_t max_row_size, double value )
{
    for(size_t k = 0; k < max_row_size; ++k)
    {
        if(vIndex[x * max_row_size + k] == y)
        {
            vValue[x * max_row_size + k] += value;
            // printf("%f \n", vValue[x * max_row_size + k]);
                k = max_row_size; // to exit for loop
            }
    }

}



__global__
void assembleGrid_GPU(
    size_t N,               // number of elements per row
    size_t dim,             // dimension
    double* l_value,        // local element's ELLPACK value vector
    size_t* l_index,        // local element's ELLPACK index vector
    size_t l_max_row_size,  // local element's ELLPACK maximum row size
    size_t l_num_rows,      // local element's ELLPACK number of rows
    double* g_value,        // global element's ELLPACK value vector
    size_t* g_index,        // global element's ELLPACK index vector
    size_t g_max_row_size,  // global element's ELLPACK maximum row size
    size_t g_num_rows,      // global element's ELLPACK number of rows
    size_t* node_index      // vector that contains the corresponding global indices of the node's local indices
)        
{
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    
    // printf("%d \n", i/2)    ;
    for ( int i = 0; i < 8; i++ )
        setAt( 2*node_index[ id/2 ] + ( id % 2 ), 2*node_index[i/2] + ( i % 2 ), g_value, g_index, g_max_row_size, valueAt( 2*(id/2) + ( id % 2 ), i, l_value, l_index, l_max_row_size) );
    
}


__global__
void assembleGrid2D_GPU(
    size_t N,               // number of elements per row
    size_t dim,             // dimension
    double* l_value,        // local element's ELLPACK value vector
    size_t* l_index,        // local element's ELLPACK index vector
    size_t l_max_row_size,  // local element's ELLPACK maximum row size
    size_t l_num_rows,      // local element's ELLPACK number of rows
    double* g_value,        // global element's ELLPACK value vector
    size_t* g_index,        // global element's ELLPACK index vector
    size_t g_max_row_size,  // global element's ELLPACK maximum row size
    size_t g_num_rows,      // global element's ELLPACK number of rows
    size_t* node_index      // vector that contains the corresponding global indices of the node's local indices
)        
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int idy = threadIdx.y + blockIdx.y*blockDim.y;

    setAt( 2*node_index[ idx/2 ] + ( idx % 2 ), 2*node_index[idy/2] + ( idy % 2 ), g_value, g_index, g_max_row_size, valueAt( 2*(idx/2) + ( idx % 2 ), 2*(idy/2) + ( idy % 2 ), l_value, l_index, l_max_row_size) );

}



// calculates the coordinates of each node
__host__
void calculateNodeCoordinates(Node* node, size_t numNodes, size_t numNodesPerDim, double h)
{
    int ycount = 0;
    for ( int i = 0 ; i < numNodes ; i += numNodesPerDim )
        {
            int count = 0;
 
            for ( int j = i ; j < numNodesPerDim + i ; j++ )
            {
                if ( j == i )
                {
                    node[j].setXCoor( 0.0 );
                    node[j].setYCoor( ycount*h);
                    count++;
                }
                
                else
                {
                    node[j].setXCoor( h*count );
                    node[j].setYCoor( ycount*h);
                    count++;
                }
            }
            ycount++;
        }
}



int main()
{

    size_t N = 2;
    size_t dim = 2;
    double rho = 0.4;

    // calculate the number of elements in the domain                                                               
    size_t numElements = pow(N,dim);
    size_t numNodesPerDim = N + 1;
    size_t numNodes = numNodesPerDim*numNodesPerDim;

    // calculate h
    float h = 1.0/N;

    // create an array of nodes
    vector<Node> node;
    
    for ( int i = 0 ; i < numNodes ; ++i )
        node.push_back(Node(i));


        calculateNodeCoordinates(&node[0], numNodes, numNodesPerDim, h);
        
        
        // creating an array of elements
        vector<Element> element;
        
        for ( int i = 0 ; i < numElements ; i++ )
        element.push_back( Element(i) );
        
        
        // adding node indices
    for ( int i = 0 ; i < numElements ; i++ )
    {
        element[i].addNode(&node[ i + i/N ]);   // lower left node
        element[i].addNode(&node[ i + i/N + 1]);   // lower right node
        element[i].addNode(&node[ i + i/N + N + 1]);   // upper left node
        element[i].addNode(&node[ i + i/N + N + 2]);   // upper right node
    }
    

    // flattened global matrix
    vector<double> K = {4,	1,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0, \
                        1,	4,	1,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0, \
                        0,	1,	8,	2,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0, \
                        0,	0,	2,	8,	1,	0,	1,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0, \
                        0,	0,	0,	1,	4,	1,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0, \
                        0,	0,	0,	0,	1,	4,	0,	0,	1,	0,	0,	0,	0,	0,	0,	0,	0,	0, \
                        0,	0,	0,	1,	0,	0,	8,	2,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0, \
                        0,	0,	0,	0,	0,	0,	2,	8,	2,	0,	0,	0,	0,	0,	0,	0,	0,	0, \
                        0,	0,	0,	0,	0,	1,	0,	2,	16,	4,	0,	0,	0,	0,	0,	0,	0,	0, \
                        0,	0,	0,	0,	0,	0,	0,	0,	4,	16,	2,	0,	1,	0,	0,	0,	0,	0, \
                        0,	0,	0,	0,	0,	0,	0,	0,	0,	2,	8,	2,	0,	0,	0,	0,	0,	0, \
                        0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	2,	8,	0,	0,	1,	0,	0,	0, \
                        0,	0,	0,	0,	0,	0,	0,	0,	0,	1,	0,	0,	4,	1,	0,	0,	0,	0, \
                        0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	1,	4,	1,	0,	0,	0, \
                        0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	1,	0,	1,	8,	2,	0,	0, \
                        0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	2,	8,	1,	0, \
                        0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	1,	4,	1, \
                        0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	0,	1,	4  };
        

    // CUDA

    // host

    size_t max_row_size = 0;
    double num_rows = 18;

    // device
    double* d_K             = nullptr;
    double* d_K_value       = nullptr;
    size_t* d_K_index       = nullptr;
    size_t* d_max_row_size  = nullptr;
    int* d_mutex            = nullptr;

    CUDA_CALL( hipMalloc( (void**)&d_K, sizeof(double) * 18 * 18 )     );
    CUDA_CALL( hipMalloc( (void**)&d_max_row_size, sizeof(size_t) )    );
    CUDA_CALL( hipMalloc( (void**)&d_mutex, sizeof(int) ) );

    CUDA_CALL( hipMemset(d_max_row_size, 0, sizeof(size_t)) );
    CUDA_CALL( hipMemset(d_mutex, 0, sizeof(int)) );
    
    CUDA_CALL( hipMemcpy(d_K, &K[0], sizeof(double) * 18 * 18 , hipMemcpyHostToDevice) ); 


    // calculate global matrix's max_row_size
    getMaxRowSize<<< 1 , 18 >>>(d_K, d_max_row_size, d_mutex, 18);
    CUDA_CALL( hipMemcpy(&max_row_size, d_max_row_size, sizeof(size_t), hipMemcpyDeviceToHost ) ); 
    
    cout << max_row_size << endl;
    // allocate device memory for global stiffness matrix's ELLPACK value and index vectors
    CUDA_CALL( hipMalloc( (void**)&d_K_value, sizeof(double) * 18 * max_row_size )     );
    CUDA_CALL( hipMalloc( (void**)&d_K_index, sizeof(size_t) * 18 * max_row_size )     );
    
    // transform K to ELLPACK
    transformToELL_GPU<<<1, 18>>>(d_K, d_K_value, d_K_index, max_row_size, 18);
    
    
    // deallocate big K matrix, no needed now
    hipFree( d_K );
    
    
    // copy and allocate the node index of each element
    
    vector<size_t*> d_node_index(numElements);
    
    for ( int i = 0 ; i < numElements ; i++ )
    {
        CUDA_CALL( hipMalloc( (void**)&d_node_index[i], sizeof(size_t) * 4 )     );
        CUDA_CALL( hipMemcpy( d_node_index[i], element[i].getNodeGlobalIndex() , sizeof(size_t) * 4 , hipMemcpyHostToDevice ) ); 
    }


    // obtain k elements' value and index vectors
    // allocate k element stiffness matrices

    vector<double*> d_ke_value(numElements);
    vector<size_t*> d_ke_index(numElements);


    // allocate and copy elements' ELLPACK stiffness matrices to device (value and index vectors)
    for ( int i = 0 ; i < numElements ; i++ )
    {
        CUDA_CALL( hipMalloc( (void**)&d_ke_value[i], sizeof(double) * 24 )     );
        CUDA_CALL( hipMalloc( (void**)&d_ke_index[i], sizeof(size_t) * 24 )     );

        CUDA_CALL( hipMemcpy( d_ke_value[i], element[i].getValueAddress() , sizeof(double) * 24 , hipMemcpyHostToDevice ) ); 
        CUDA_CALL( hipMemcpy( d_ke_index[i], element[i].getIndexAddress() , sizeof(size_t) * 24 , hipMemcpyHostToDevice ) ); 
    }
    


    // array of the initial design variable

    vector<double> design(numElements);

    for ( int i = 0 ; i < numElements ; i++ )
        design.push_back(rho);

    double* d_design = nullptr;
        
    CUDA_CALL( hipMalloc( (void**)&d_design, sizeof(double) * numElements )     );
    CUDA_CALL( hipMemcpy( d_design, &design[0] , sizeof(double) * numElements , hipMemcpyHostToDevice ) ); 
    


    // allocate and copy the empty global matrix

    Element global;
    
    double* d_KG_value;
    size_t* d_KG_index;
    
    CUDA_CALL( hipMalloc( (void**)&d_KG_value, sizeof(double) * 72 )     );
    CUDA_CALL( hipMalloc( (void**)&d_KG_index, sizeof(size_t) * 72 )     );
    CUDA_CALL( hipMemcpy( d_KG_value, global.getValueAddress() , sizeof(double) * 72 , hipMemcpyHostToDevice ) ); 
    CUDA_CALL( hipMemcpy( d_KG_index, global.getIndexAddress() , sizeof(size_t) * 72 , hipMemcpyHostToDevice ) ); 
    

    // add local stiffness matrices into the global

    // for ( int i = 0 ; i < numElements ; i++ )
    // {
    //     assembleGrid_GPU<<<1, 8>>>( 2, 2, d_ke_value[i], d_ke_index[i], element[i].max_row_size(), element[i].num_rows(), d_KG_value, d_K_index, global.max_row_size(), global.num_rows(), d_node_index[i] );
    //     hipDeviceSynchronize();
    // }

    dim3 blockDim(8,8,1);
    //     assembleGrid2D_GPU<<<1, blockDim>>>( 2, 2, d_ke_value[0], d_ke_index[0], element[0].max_row_size(), element[0].num_rows(), d_KG_value, d_K_index, global.max_row_size(), global.num_rows(), d_node_index[0] );

    for ( int i = 0 ; i < numElements ; i++ )
        {
            assembleGrid2D_GPU<<<1, blockDim>>>( 2, 2, d_ke_value[i], d_ke_index[i], element[i].max_row_size(), element[i].num_rows(), d_KG_value, d_K_index, global.max_row_size(), global.num_rows(), d_node_index[i] );
            hipDeviceSynchronize();
        }



    printVector_GPU<<<1,72>>> ( d_KG_value, 72 );
    // printVector_GPU<<<1,72>>> ( d_K_index, 72 );
    hipDeviceSynchronize();

}
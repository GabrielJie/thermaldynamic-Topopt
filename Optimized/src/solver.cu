#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/solver.h"
#include "../include/cudakernels.h"

using namespace std;

Solver::Solver(vector<double*> d_value, vector<size_t*> d_index, vector<double*> d_p_value, vector<size_t*> d_p_index, size_t numLevels, vector<size_t> num_rows, vector<size_t> max_row_size, vector<size_t> p_max_row_size, double damp)
: m_d_value(d_value), m_d_index(d_index), m_d_p_value(d_p_value), m_d_p_index(d_p_index), m_numLevels(numLevels), m_num_rows(num_rows), m_max_row_size(max_row_size), m_p_max_row_size(p_max_row_size), m_damp(damp) 
{
    

}

void Solver::set_verbose(bool verbose, bool bs_verbose) { m_verbose = verbose; m_bs_verbose = bs_verbose; }

// DEBUG:
void Solver::set_steps(size_t step, size_t bs_step)
{
    m_step = step;
    m_bs_step = bs_step;
}


void Solver::set_num_prepostsmooth(size_t pre_n, size_t post_n)
{
    m_numPreSmooth = pre_n;
    m_numPostSmooth = post_n;
}

void Solver::set_convergence_params( size_t maxIter, double minRes, double minRed )
{
	m_maxIter = maxIter;
	m_minRes = minRes;
	m_minRed = minRed;
}


void Solver::set_convergence_params_( size_t maxIter, size_t bs_maxIter, double minRes, double minRed )
{
	m_maxIter = maxIter;
	m_bs_maxIter = bs_maxIter;
	m_minRes = minRes;
	m_minRed = minRed;
}


void Solver::set_bs_convergence_params( size_t maxIter, double minRes, double minRed )
{
	m_bs_maxIter = maxIter;
	m_bs_minRes = minRes;
	m_bs_minRed = minRed;
}



// TODO: could try as a destructor
// void Solver::deallocate()
Solver::~Solver()
{
    // // cout << "solver : deallocate" << endl;
    // CUDA_CALL( hipFree(m_d_res0) );
    // CUDA_CALL( hipFree(m_d_res) );
    // CUDA_CALL( hipFree(m_d_lastRes) );
    // CUDA_CALL( hipFree(m_d_minRes) );
    // CUDA_CALL( hipFree(m_d_minRed) );
    // CUDA_CALL( hipFree(m_d_r) );
    // CUDA_CALL( hipFree(m_d_c) );
    // CUDA_CALL( hipFree(m_d_step) );
    // CUDA_CALL( hipFree(m_d_bs_step) );
    
    // // base solver
    // CUDA_CALL( hipFree(m_d_bs_r) );
    // CUDA_CALL( hipFree(m_d_bs_z) );
    // CUDA_CALL( hipFree(m_d_bs_res) );
    // CUDA_CALL( hipFree(m_d_bs_lastRes) );
    // CUDA_CALL( hipFree(m_d_bs_res0) );
    // // CUDA_CALL( hipFree(m_d_bs_minRes) );
    // // CUDA_CALL( hipFree(m_d_bs_minRed) );
    // CUDA_CALL( hipFree(m_d_bs_rho_old) );
    // CUDA_CALL( hipFree(m_d_bs_p) );
    // CUDA_CALL( hipFree(m_d_bs_alpha) );
    // CUDA_CALL( hipFree(m_d_bs_alpha_temp) );
    
}




void Solver::set_cycle(const char type)
{
    switch(type){
        case 'V': m_gamma = 1; break;
        case 'W': m_gamma = 2; break;
        case 'F': m_gamma = -1; break;
        
        default: std::cout << "Cycle type '" << type << "' invalid argument" << std::endl;
        throw std::invalid_argument("Cycle type: invalid argument");
    }
}
    
bool Solver::init()
{

        m_topLev = m_numLevels - 1;

        // convergence checks
        CUDA_CALL( hipMalloc((void**)&m_d_foo, sizeof(bool)) );
        CUDA_CALL( hipMemcpy(m_d_foo, &m_foo, sizeof(bool), hipMemcpyHostToDevice) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_foo, sizeof(bool)) );
        CUDA_CALL( hipMemcpy(m_d_bs_foo, &m_bs_foo, sizeof(bool), hipMemcpyHostToDevice) );
        


        // calculate cuda grid and block dimensions of each level
        m_gridDim.resize(m_numLevels);
        m_blockDim.resize(m_numLevels);

        for ( int i = 0 ; i < m_numLevels ; i++ )
            calculateDimensions(m_num_rows[i], m_gridDim[i], m_blockDim[i]);
		    
		m_gridDim_cols.resize(m_numLevels - 1);
        m_blockDim_cols.resize(m_numLevels - 1);

        for ( int i = 0 ; i < m_numLevels - 1; i++ )
		    calculateDimensions(m_num_rows[i], m_gridDim_cols[i], m_blockDim_cols[i]);

        

        CUDA_CALL( hipMalloc((void**)&m_d_r, sizeof(double) * m_num_rows[m_topLev]) );
        CUDA_CALL( hipMemset(m_d_r, 0, sizeof(double) * m_num_rows[m_topLev]) );
        CUDA_CALL( hipMalloc((void**)&m_d_c, sizeof(double) * m_num_rows[m_topLev]) );
        CUDA_CALL( hipMemset(m_d_c, 0, sizeof(double) * m_num_rows[m_topLev]) );

        // TODO: perhaps you could use a temp variable here, no need to malloc?
        // temp residuum
        CUDA_CALL( hipMalloc((void**)&m_d_res0, sizeof(double)) );
        CUDA_CALL( hipMemset(m_d_res0, 0, sizeof(double)) );

        // last residuum
        CUDA_CALL( hipMalloc((void**)&m_d_lastRes, sizeof(double)) );
        CUDA_CALL( hipMemset(m_d_lastRes, 0, sizeof(double)) );
        
        // current residuum
        CUDA_CALL( hipMalloc((void**)&m_d_res, sizeof(double)) );
        CUDA_CALL( hipMemset(m_d_res, 0, sizeof(double)) );
    
        // minimum required residuum for convergence
        // d_m_minRes;
        CUDA_CALL( hipMalloc((void**)&m_d_minRes, sizeof(double)) );
        CUDA_CALL( hipMemcpy(m_d_minRes, &m_minRes, sizeof(double), hipMemcpyHostToDevice) );
        
        // minimum required reduction for convergence
        // d_m_minRed;
        CUDA_CALL( hipMalloc((void**)&m_d_minRed, sizeof(double)) );
        CUDA_CALL( hipMemcpy(m_d_minRed, &m_minRed, sizeof(double), hipMemcpyHostToDevice) );
        
        // steps
        CUDA_CALL( hipMalloc((void**)&m_d_step, sizeof(size_t)) );
        CUDA_CALL( hipMemset(m_d_step, 0, sizeof(size_t)) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_step, sizeof(size_t)) );
        CUDA_CALL( hipMemset(m_d_bs_step, 0, sizeof(size_t)) );

        /// GMG precond
        // residuum and correction vectors on each level
        m_d_gmg_r.resize(m_numLevels);
        m_d_gmg_c.resize(m_numLevels);
        
        // temporary residuum vectors for GMG
        m_d_rtmp.resize(m_numLevels);

        // temporary correction vectors for GMG
        m_d_ctmp.resize(m_numLevels);

        for ( int i = 0 ; i < m_numLevels ; i++ )
        {
            CUDA_CALL( hipMalloc((void**)&m_d_gmg_r[i], sizeof(double) * m_num_rows[i] ) );
            CUDA_CALL( hipMalloc((void**)&m_d_gmg_c[i], sizeof(double) * m_num_rows[i] ) );
            CUDA_CALL( hipMemset(m_d_gmg_r[i], 0, sizeof(double) * m_num_rows[i] ) );
            CUDA_CALL( hipMemset(m_d_gmg_c[i], 0, sizeof(double) * m_num_rows[i] ) );

            CUDA_CALL( hipMalloc((void**)&m_d_rtmp[i], sizeof(double) * m_num_rows[i] ) );
            CUDA_CALL( hipMemset(m_d_rtmp[i], 0, sizeof(double) * m_num_rows[i] ) );

            CUDA_CALL( hipMalloc((void**)&m_d_ctmp[i], sizeof(double) * m_num_rows[i] ) );
            CUDA_CALL( hipMemset(m_d_ctmp[i], 0, sizeof(double) * m_num_rows[i] ) );
        }


        // base-solver

        CUDA_CALL( hipMalloc((void**)&m_d_bs_r, sizeof(double) * m_num_rows[0] ) );
        CUDA_CALL( hipMemset(m_d_bs_r, 0, sizeof(double) * m_num_rows[0] ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_z, sizeof(double) * m_num_rows[0] ) );
        CUDA_CALL( hipMemset(m_d_bs_z, 0, sizeof(double) * m_num_rows[0] ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_p, sizeof(double) * m_num_rows[0] ) );
        CUDA_CALL( hipMemset(m_d_bs_p, 0, sizeof(double) * m_num_rows[0] ) );

        CUDA_CALL( hipMalloc((void**)&m_d_bs_res, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_res, 0, sizeof(double) ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_res0, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_res0, 0, sizeof(double) ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_lastRes, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_lastRes, 0, sizeof(double) ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_rho, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_rho, 0, sizeof(double) ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_rho_old, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_rho_old, 0, sizeof(double) ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_alpha, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_alpha, 0, sizeof(double) ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_alpha_temp, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_alpha_temp, 0, sizeof(double) ) );
        
        // TODO: might not be needed
        // CUDA_CALL( hipMalloc((void**)&m_d_bs_minRed, sizeof(double) ) );
        // CUDA_CALL( hipMemcpy(m_d_bs_minRed, &m_minRed, sizeof(double), hipMemcpyHostToDevice) );
        // CUDA_CALL( hipMalloc((void**)&m_d_bs_minRes, sizeof(double) ) );
        // CUDA_CALL( hipMemcpy(m_d_bs_minRes, &m_minRes, sizeof(double), hipMemcpyHostToDevice) );
       

    return true;
}

bool Solver::reinit()
{
        
        setToZero<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>( m_d_r, m_num_rows[m_topLev] );
        setToZero<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>( m_d_c, m_num_rows[m_topLev] );

        for ( int lev = 0 ; lev < m_numLevels ; lev++ )
        {
            setToZero<<<m_gridDim[lev], m_blockDim[lev]>>>( m_d_gmg_r[lev], m_num_rows[lev] );
            setToZero<<<m_gridDim[lev], m_blockDim[lev]>>>( m_d_gmg_c[lev], m_num_rows[lev] );
            setToZero<<<m_gridDim[lev], m_blockDim[lev]>>>( m_d_rtmp[lev], m_num_rows[lev] );
            setToZero<<<m_gridDim[lev], m_blockDim[lev]>>>( m_d_ctmp[lev], m_num_rows[lev] );
        }

        // scalars
        setToZero<<<1, 1>>>( m_d_res0, 1 );
        setToZero<<<1, 1>>>( m_d_lastRes, 1 );
        setToZero<<<1, 1>>>( m_d_res, 1 );
        setToZero<<<1, 1>>>( m_d_step, 1 );
        setToZero<<<1, 1>>>( m_d_bs_step, 1 );


        // base-solver
        setToZero<<<m_gridDim[0], m_blockDim[0]>>>( m_d_bs_r, m_num_rows[0] );
        setToZero<<<m_gridDim[0], m_blockDim[0]>>>( m_d_bs_z, m_num_rows[0] );
        setToZero<<<m_gridDim[0], m_blockDim[0]>>>( m_d_bs_p, m_num_rows[0] );
        setToZero<<<1, 1>>>( m_d_bs_res, 1 );
        setToZero<<<1, 1>>>( m_d_bs_res0, 1 );
        setToZero<<<1, 1>>>( m_d_bs_lastRes, 1 );
        setToZero<<<1, 1>>>( m_d_bs_rho, 1 );
        setToZero<<<1, 1>>>( m_d_bs_rho_old, 1 );
        setToZero<<<1, 1>>>( m_d_bs_alpha, 1 );
        setToZero<<<1, 1>>>( m_d_bs_alpha_temp, 1 );

        return true;
}

bool Solver::precond(double* m_d_c, double* m_d_r)
{
    // cout << "precond" << endl;
    hipDeviceSynchronize();


    // reset correction
    // c.resize(d.size()); 
    // c = 0.0;
	setToZero<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_c, m_num_rows[m_topLev]);

    // Vector<double> rtmp(r);
	vectorEquals_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_rtmp[m_topLev], m_d_r, m_num_rows[m_topLev]);
    
	// NOTE: the original d_c and d_r from i_s.cu stay here
	// d_gmg_c[topLev] = d_c
	// d_gmg_r[topLev] = d_r
	vectorEquals_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_gmg_c[m_topLev], m_d_c, m_num_rows[m_topLev]);
	hipDeviceSynchronize();
	vectorEquals_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_gmg_r[m_topLev], m_d_r, m_num_rows[m_topLev]);
	hipDeviceSynchronize();

    precond_add_update_GPU(m_d_gmg_c[m_topLev], m_d_rtmp[m_topLev], m_topLev, m_gamma);

    vectorEquals_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_c, m_d_gmg_c[m_topLev], m_num_rows[m_topLev]);
	// hipDeviceSynchronize();
	vectorEquals_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_r, m_d_gmg_r[m_topLev], m_num_rows[m_topLev]);

    return true;
}

// A*c = r ==> A_coarse*d_bs_u = d_bs_b
bool Solver::base_solve(double* d_bs_u, double* d_bs_b)
{

    // resetting base solver variables to zero
    setToZero<<<1,m_num_rows[0]>>>(m_d_bs_r, m_num_rows[0]);
    setToZero<<<1,m_num_rows[0]>>>(m_d_bs_p, m_num_rows[0]);
    setToZero<<<1,m_num_rows[0]>>>(m_d_bs_z, m_num_rows[0]);
    setToZero<<<1,1>>>(m_d_bs_rho, 1);
    setToZero<<<1,1>>>(m_d_bs_rho_old, 1);
    setToZero<<<1,1>>>(m_d_bs_alpha, 1);
    setToZero<<<1,1>>>(m_d_bs_alpha_temp, 1);
    setToZero<<<1,1>>>(m_d_bs_res, 1);
    setToZero<<<1,1>>>(m_d_bs_res0, 1);
    setToZero<<<1,1>>>(m_d_bs_lastRes, 1);
    setToZero<<<1,1>>>(m_d_bs_step, 1);
    setToTrue<<<1,1>>>(m_d_bs_foo);
    hipDeviceSynchronize();

    
    m_bs_foo = true;
    // hipDeviceSynchronize();
    // cout << "d_bs_b\n";
    // printVector_GPU<<<1,m_num_rows[0]>>>( d_bs_b, m_num_rows[0] );
    // hipDeviceSynchronize();


    // if (m_bs_verbose)
        // printELLrow(0, m_d_value[0], m_d_index[0], m_max_row_size[0], m_num_rows[0], m_num_rows[0]);








    // m_d_bs_r = d_bs_b - A*d_bs_u
    ComputeResiduum_GPU<<<m_gridDim[0],m_blockDim[0]>>>(m_num_rows[0], m_max_row_size[0], m_d_value[0], m_d_index[0], d_bs_u, m_d_bs_r, d_bs_b);

    // norm_GPU(m_d_bs_res, m_d_bs_r, m_num_rows[0], m_gridDim[0], m_blockDim[0]);
    norm_GPU<<<m_gridDim[0], m_blockDim[0]>>>(m_d_bs_res, m_d_bs_r, m_num_rows[0]);
    //     if ( m_bs_verbose )
    //     print_GPU<<<1,1>>>( m_d_bs_res );
    hipDeviceSynchronize();


    equals_GPU<<<1,1>>>(m_d_bs_res0, m_d_bs_res);
    
    if ( m_bs_verbose )
    {
        // cout << "\n";
        // cout << "## CG  ##################################################################" << endl;
        // cout << "  Iter     Residuum       Required       Rate        Reduction     Required" << endl;
        cout << "CG  : ";
        hipDeviceSynchronize();
        printInitialResult_GPU<<<1,1>>>(m_d_bs_res0, m_d_minRes, m_d_minRed);
        hipDeviceSynchronize();
    }
	
    // hipDeviceSynchronize();
    // printVector_GPU<<<1,m_num_rows[0]>>>( d_bs_u, m_num_rows[0] );
    // hipDeviceSynchronize();
	
 
    // check iteration conditions before the loop

    checkIterationConditions<<<1,1>>>(m_d_bs_foo, m_d_bs_step, m_d_bs_res, m_d_bs_res0, m_d_minRes, m_d_minRed, m_bs_maxIter);
    CUDA_CALL( hipMemcpy( &m_bs_foo, m_d_bs_foo, sizeof(bool), hipMemcpyDeviceToHost) 	);
    
    // //DEBUG:
    // cout << "CG checkiteration. m_bs_foo = " << m_bs_foo << endl;
    
    if (!m_bs_foo) return true;

    else
    {
   
    addStep<<<1,1>>>(m_d_bs_step);

    // TODO: add this before foo loop
    // checkIterationConditions<<<1,1>>>(d_cg_foo, d_cg_step, d_cg_res, d_cg_res0, d_cg_m_minRes, d_cg_m_minRed, d_cg_m_maxIter);
    

    // foo loop
    int bs_step = 1;
    while(m_bs_foo || bs_step < m_bs_maxIter)
    {

    
    
    // while(bs_step < m_bs_step)
    // TODO: check
    // smoother( m_d_bs_z, m_d_bs_r, 0);
    
    // cout << "iteration " << bs_step << endl;
    // hipDeviceSynchronize();
    // print_GPU<<<1,1>>>( m_d_bs_step );
    // hipDeviceSynchronize();

  




    // z = r
    vectorEquals_GPU<<<m_gridDim[0],m_blockDim[0]>>>(m_d_bs_z, m_d_bs_r, m_num_rows[0]);

    // if ( bs_step == 1 )
    // {
    //     print_GPU<<<1,1>>>( m_d_bs_res );
    //     // printVector_GPU<<<1,m_num_rows[0]>>>( m_d_bs_z, m_num_rows[0] );
    //     // printVector_GPU<<<1,m_num_rows[0]>>>( m_d_bs_r, m_num_rows[0] );
    //     hipDeviceSynchronize();
    // }
    


    // hipDeviceSynchronize();
    // printVector_GPU<<<1,m_num_rows[0]>>>( m_d_bs_z, m_num_rows[0] );
    // hipDeviceSynchronize();


    // rho = < z, r >
    dotProduct(m_d_bs_rho, m_d_bs_r, m_d_bs_z, m_num_rows[0], m_gridDim[0], m_blockDim[0]);
    
    // if ( bs_step == 1 )
    // {
    //     print_GPU<<<1,1>>>( m_d_bs_rho );
    //     // printVector_GPU<<<1,m_num_rows[0]>>>( m_d_bs_z, m_num_rows[0] );
    //     hipDeviceSynchronize();
    // }



    hipDeviceSynchronize();

    // // DEBUG:
    // if ( m_bs_verbose )
    //     print_GPU<<<1,1>>>( m_d_bs_rho );
    // hipDeviceSynchronize();


    calculateDirectionVector<<<m_gridDim[0],m_blockDim[0]>>>(m_d_bs_step, m_d_bs_p, m_d_bs_z, m_d_bs_rho, m_d_bs_rho_old, m_num_rows[0]);
    
        
    // if ( bs_step == 1 )
    // {
    //     // print_GPU<<<1,1>>>( m_d_bs_rho );
    //     // printVector_GPU<<<1,m_num_rows[0]>>>( m_d_bs_z, m_num_rows[0] );
    //     printVector_GPU<<<1,m_num_rows[0]>>>( m_d_bs_p, m_num_rows[0] );
    //     hipDeviceSynchronize();
    // }




    // hipDeviceSynchronize();
    // printVector_GPU<<<1,m_num_rows[0]>>>( m_d_bs_p, m_num_rows[0] );
    // hipDeviceSynchronize();

    /// z = A*p
    Apply_GPU<<<m_gridDim[0],m_blockDim[0]>>>( m_num_rows[0], m_max_row_size[0], m_d_value[0], m_d_index[0], m_d_bs_p, m_d_bs_z );

      
    // if ( bs_step == 1 )
    // {
    //     // print_GPU<<<1,1>>>( m_d_bs_rho );
    //     printVector_GPU<<<1,m_num_rows[0]>>>( m_d_bs_z, m_num_rows[0] );
    //     hipDeviceSynchronize();
    // }
    

    hipDeviceSynchronize();
    // printVector_GPU<<<1,m_num_rows[0]>>>( m_d_bs_z, m_num_rows[0] );
    // printELL_GPU<<<1,1>>>( m_d_value[0], m_d_index[0], m_max_row_size[0], m_num_rows[0], m_num_rows[0]);
    hipDeviceSynchronize();

    // alpha = rho / (p * z)
    calculateAlpha(m_d_bs_alpha, m_d_bs_rho, m_d_bs_p, m_d_bs_z, m_d_bs_alpha_temp, m_num_rows[0], m_gridDim[0], m_blockDim[0] );

     
    // if ( bs_step == 4 )
    // {
    //     print_GPU<<<1,1>>>( m_d_bs_alpha );
    //     // printVector_GPU<<<1,m_num_rows[0]>>>( m_d_bs_z, m_num_rows[0] );
    //     hipDeviceSynchronize();
    // }

    // // //DEBUG:
    // if ( m_bs_verbose )
    // {
    //     hipDeviceSynchronize();
        // print_GPU<<<1,1>>>( m_d_bs_alpha );
    //     // printVector_GPU<<<m_gridDim[0],m_blockDim[0]>>>(m_d_bs_z, m_num_rows[0]);
    //     // printELLrow(0, m_d_value[0], m_d_index[0], m_max_row_size[0], m_num_rows[0], m_num_rows[0]);
    //     hipDeviceSynchronize();
    // }

        hipDeviceSynchronize();

    // add correction to solution
    // u = u + alpha * p
    axpy_GPU<<<m_gridDim[0],m_blockDim[0]>>>(d_bs_u, m_d_bs_alpha, m_d_bs_p, m_num_rows[0]);
    hipDeviceSynchronize();

       
    // if ( bs_step == 4 )
    // {
    //     // print_GPU<<<1,1>>>( m_d_bs_alpha );
    //     printVector_GPU<<<1,m_num_rows[0]>>>( d_bs_u, m_num_rows[0] );
    //     hipDeviceSynchronize();
    // }


    // update residuum
    // r = r - alpha * z
    axpy_neg_GPU<<<m_gridDim[0],m_blockDim[0]>>>(m_d_bs_r, m_d_bs_alpha, m_d_bs_z, m_num_rows[0]);
    hipDeviceSynchronize();
    
    // printVector_GPU<<<m_gridDim[0],m_blockDim[0]>>>(m_d_bs_r, m_num_rows[0]);
    // hipDeviceSynchronize();

  



    // compute residuum
    // lastRes = res;
    equals_GPU<<<1,1>>>(m_d_bs_lastRes, m_d_bs_res);
        hipDeviceSynchronize();
    


    

    // res = r.norm();
    norm_GPU(m_d_bs_res, m_d_bs_r, m_num_rows[0], m_gridDim[0], m_blockDim[0]);
        hipDeviceSynchronize();

   

    // print_GPU<<<1,1>>>( m_d_bs_res );
    // store old rho
    // rho_old = rho;
    vectorEquals_GPU<<<m_gridDim[0],m_blockDim[0]>>>(m_d_bs_rho_old, m_d_bs_rho, m_num_rows[0]);
    hipDeviceSynchronize();

    if ( m_bs_verbose )
    {
    cout << "CG  : ";
    hipDeviceSynchronize();
    printResult_GPU<<<1,1>>>(m_d_bs_step, m_d_bs_res, m_d_minRes, m_d_bs_lastRes, m_d_bs_res0, m_d_minRed);
    hipDeviceSynchronize();
    }
       
    
    // checkIterationConditions<<<1,1>>>(m_d_bs_foo, m_d_bs_step, m_d_bs_res, m_d_bs_res0, m_bs_minRes, m_d_minRed, m_bs_maxIter);
    // checkIterationConditions<<<1,1>>>(m_d_bs_foo, m_d_bs_step, m_d_bs_res, m_d_bs_res0, m_d_minRes, m_d_minRed, m_bs_maxIter);
    checkIterationConditionsBS<<<1,1>>>(m_d_bs_foo, m_d_bs_step, m_bs_maxIter, m_d_bs_res, m_d_minRes);
    
    CUDA_CALL( hipMemcpy( &m_bs_foo, m_d_bs_foo, sizeof(bool), hipMemcpyDeviceToHost) 	);
    
    // //DEBUG:
    // cout << "CG checkiteration. m_bs_foo = " << m_bs_foo << endl;
    
    if (!m_bs_foo) break;

    addStep<<<1,1>>>(m_d_bs_step);
    
    // DEBUG:
    bs_step++;
    hipDeviceSynchronize();
    }
    
    // if ( m_bs_verbose )
    //     cout << "\n";

    return true;
    
    }
}


bool Solver::precond_add_update_GPU(double* d_c, double* d_r, std::size_t lev, int cycle)
{
    // cout << "precond_add_update" << endl;

    // std::cout <<"gmg.cu : setToZero()" << std::endl;
    // Vector<double> ctmp(c.size(), 0.0, c.layouts());
    setToZero<<< m_gridDim[lev], m_blockDim[lev] >>>( m_d_ctmp[lev], m_num_rows[lev] );			
    hipDeviceSynchronize();

    
    // static int apsie = 0;
    // if (apsie == 0)
    // {
    // hipDeviceSynchronize();
    // printVector_GPU<<<1,450>>>( d_r, 450 );
    // hipDeviceSynchronize();
    // cout << "\n";
    // cout << "\n";
    // }

    // if on base level
	if( lev == 0 )
	{
        // cout << "base level" << endl;
        base_solve(m_d_ctmp[lev], d_r);
        hipDeviceSynchronize();

    // // DEBUG:
    //     hipDeviceSynchronize();
        // printVector_GPU<<<1,8>>>( m_d_ctmp[lev], 8 );
        // hipDeviceSynchronize();
        

        // c += ctmp;
		addVector_GPU<<< m_gridDim[lev], m_blockDim[lev] >>>(d_c, m_d_ctmp[lev], m_num_rows[0]);
        // r -= A[0] * c;
        hipDeviceSynchronize();

        //CHECK:
        // r = r - A[0] * ctmp0
		UpdateResiduum_GPU<<< m_gridDim[lev], m_blockDim[lev] >>>(m_num_rows[lev], m_max_row_size[lev], m_d_value[lev], m_d_index[lev], m_d_ctmp[lev], d_r);
		hipDeviceSynchronize();

        // hipDeviceSynchronize();
        // printVector_GPU<<<1,m_num_rows[lev]>>> ( d_r, m_num_rows[lev] );
        // hipDeviceSynchronize();


        return true;
    }

    // presmooth
    
    for ( int i = 0 ; i < m_numPreSmooth ; i++)
    {
        smoother( m_d_ctmp[lev], d_r, lev );
        hipDeviceSynchronize();
        // hipDeviceSynchronize();
        // printVector_GPU<<<1,18>>>( m_d_ctmp[lev], 18 );
        // hipDeviceSynchronize();


        // c += ctmp;
        addVector_GPU<<<m_gridDim[lev], m_blockDim[lev]>>>( d_c, m_d_ctmp[lev], m_num_rows[lev] );
        
        
        // r -= A[lev] * ctmp;
        UpdateResiduum_GPU<<< m_gridDim[lev], m_blockDim[lev] >>>(m_num_rows[lev], m_max_row_size[lev], m_d_value[lev], m_d_index[lev], m_d_ctmp[lev], d_r);
        hipDeviceSynchronize();
    }

    
    // restrict defect
    setToZero<<<m_gridDim_cols[lev-1],m_blockDim_cols[lev-1]>>>( m_d_gmg_r[lev-1], m_num_rows[lev-1] );

    // hipDeviceSynchronize();
    // printELL_GPU<<<1,1>>>(m_d_p_value[0], m_d_p_index[0], m_p_max_row_size[0], m_num_rows[1], m_num_rows[0]);
    // hipDeviceSynchronize();

//    hipDeviceSynchronize();
//     cout << "aps" << endl;
//     hipDeviceSynchronize();


    
    // TODO: r_coarse = R * r
    /// r_coarse = P^T * r
    ApplyTransposed_GPU<<<m_gridDim[lev],m_blockDim[lev]>>>(m_num_rows[lev], m_p_max_row_size[lev-1], m_d_p_value[lev-1], m_d_p_index[lev-1], d_r, m_d_gmg_r[lev-1]);
    hipDeviceSynchronize();
    setToZero<<<m_gridDim_cols[lev-1],m_blockDim_cols[lev-1]>>>( m_d_gmg_c[lev-1], m_num_rows[lev-1] );


    
    if(cycle == -1) // F-cycle
	{

		// cout << "F cycle" << endl; // DEBUG:
		// hipDeviceSynchronize();
			// one F-Cycle ...
		    if( !precond_add_update_GPU(m_d_ctmp[lev-1], m_d_rtmp[lev-1], lev-1, -1) )  // TODO: check ctmp or gmg_c?
			{
		        std::cout << "gmg failed on level " << lev << ". Aborting." << std::endl;
		        return false;
		    }

		    // ... followed by a V-Cycle
		   	if( !precond_add_update_GPU(m_d_ctmp[lev-1], m_d_rtmp[lev-1], lev-1, 1) )
			{
		        std::cout << "gmg failed on level " << lev << ". Aborting." << std::endl;
		        return false;
			}
	}

    else
	{

		// V- and W-cycle
		for (int g = 0; g < cycle; ++g)
		{

			if( !precond_add_update_GPU(m_d_gmg_c[lev-1], m_d_gmg_r[lev-1], lev-1, cycle) )
			{
				std::cout << "gmg failed on level " << lev << ". Aborting." << std::endl;
				return false;
			}
		
		}
    }

    // DEBUG:
        // hipDeviceSynchronize();
        // printVector_GPU<<<1,8>>>( m_d_gmg_r[lev-1], 8 );
        // hipDeviceSynchronize();
    // hipDeviceSynchronize();
    // printELL_GPU<<<1,1>>>(m_d_value[0], m_d_index[0], m_max_row_size[0], m_num_rows[0], m_num_rows[0]);
    // hipDeviceSynchronize();
    
    /// prolongate coarse grid correction
	// ctmp = P[lev-1] * c_coarse;
    Apply_GPU<<<m_gridDim[lev],m_blockDim[lev]>>>( m_num_rows[lev], m_p_max_row_size[lev-1], m_d_p_value[lev-1], m_d_p_index[lev-1], m_d_gmg_c[lev-1], m_d_ctmp[lev]);
    hipDeviceSynchronize();
    /// add correction and update defect
	// c += ctmp;
	addVector_GPU<<<m_gridDim[lev],m_blockDim[lev]>>>(d_c, m_d_ctmp[lev], m_num_rows[lev]);
    hipDeviceSynchronize();
    UpdateResiduum_GPU<<<m_gridDim[lev],m_blockDim[lev]>>>( m_num_rows[lev], m_max_row_size[lev] , m_d_value[lev], m_d_index[lev], m_d_ctmp[lev], d_r);
    hipDeviceSynchronize();
    
    // postsmooth
    for ( int i = 0 ; i < m_numPostSmooth ; i++)
    {
        smoother( m_d_ctmp[lev], d_r, lev );

         // c += ctmp;
        addVector_GPU<<<m_gridDim[lev], m_blockDim[lev]>>>( d_c, m_d_ctmp[lev], m_num_rows[lev] );

        UpdateResiduum_GPU<<< m_gridDim[lev], m_blockDim[lev] >>>(m_num_rows[lev], m_max_row_size[lev], m_d_value[lev], m_d_index[lev], m_d_ctmp[lev], d_r);

    }

    
    // static int apsie = 0;
    // if (apsie == 0)
    // {
    // hipDeviceSynchronize();
    // printVector_GPU<<<1,450>>>( d_r, 450 );
    // hipDeviceSynchronize();
    // cout << "\n";
    // cout << "\n";
    // }
    // apsie++;

    hipDeviceSynchronize();
    return true;
}

bool Solver::smoother(double* d_c, double* d_r, int lev)
{
    
    // cout << "smoother" << endl;
    
    Jacobi_Precond_GPU<<<m_gridDim[lev], m_blockDim[lev]>>>(d_c, m_d_value[lev], m_d_index[lev], m_max_row_size[lev], d_r, m_num_rows[lev], m_damp);

    return true;
}




bool Solver::solve(double* d_u, double* d_b, vector<double*> d_value)
{
       

    // // DEBUG:
    // printVector_GPU<<<1, m_num_rows[m_topLev]>>> ( d_b, m_num_rows[m_topLev]);
    // hipDeviceSynchronize();


    //TODO: cantikkan
    setToZero<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>( d_u, m_num_rows[m_topLev] );
    setToTrue<<<1,1>>>(m_d_foo);
    m_foo = true;

    m_d_value = d_value;
    

    // r = b - A*u
    ComputeResiduum_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_num_rows[m_topLev], m_max_row_size[m_topLev], m_d_value[m_topLev], m_d_index[m_topLev], d_u, m_d_r, d_b);
    hipDeviceSynchronize();
    
    
    // d_res0 = norm(m_d_r)
    // norm_GPU(m_d_res0, m_d_r, m_num_rows[m_topLev], m_gridDim[m_topLev], m_blockDim[m_topLev]);
    norm_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_res0, m_d_r, m_num_rows[m_topLev]);
    hipDeviceSynchronize();

    // res = res0;
    equals_GPU<<<1,1>>>(m_d_res, m_d_res0);	

    if ( m_verbose )
    {
        cout << "GMG : ";
        hipDeviceSynchronize();
        printInitialResult_GPU<<<1,1>>>(m_d_res0, m_d_minRes, m_d_minRed);
        hipDeviceSynchronize();
    }

    addStep<<<1,1>>>(m_d_step);

    // foo loop
    
    // DEBUG:
    // int step = 0;
    // cout << "step = " << step << endl;
    // while(m_foo || step < m_step)

    
    while(m_foo)
    {
        
    precond(m_d_c, m_d_r);
    hipDeviceSynchronize();
    // add correction to solution
    // u += c;
    addVector_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>( d_u, m_d_c, m_num_rows[m_topLev] );
    hipDeviceSynchronize();

    // // DEBUG:
    // // printVector_GPU<<<1,18>>>( m_d_c, 18 );
    // // hipDeviceSynchronize();


    // update residuum r = r - A*c
    UpdateResiduum_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>( m_num_rows[m_topLev], m_max_row_size[m_topLev], m_d_value[m_topLev], m_d_index[m_topLev], m_d_c, m_d_r );
    hipDeviceSynchronize();
    

    // remember last residuum norm
    // lastRes = res;
    equals_GPU<<<1,1>>>(m_d_lastRes, m_d_res);
    hipDeviceSynchronize();

    // compute new residuum norm
    // res = r.norm();
    // norm_GPU<<<gridDim,blockDim>>>(d_res, d_r, A.num_rows());
    // TODO:
    norm_GPU(m_d_res, m_d_r, m_num_rows[m_topLev], m_gridDim[m_topLev], m_blockDim[m_topLev]);
    hipDeviceSynchronize();

    if ( m_verbose )
    {
    cout << "GMG : ";
    hipDeviceSynchronize();
    printResult_GPU<<<1,1>>>(m_d_step, m_d_res, m_d_minRes, m_d_lastRes, m_d_res0, m_d_minRed);
    hipDeviceSynchronize();
    }

    checkIterationConditions<<<1,1>>>(m_d_foo, m_d_step, m_d_res, m_d_res0, m_d_minRes, m_d_minRed, m_maxIter);
    CUDA_CALL( hipMemcpy( &m_foo, m_d_foo, sizeof(bool), hipMemcpyDeviceToHost) 	);
    
    // if ( !m_foo )
    //     break;

    addStep<<<1,1>>>(m_d_step);
    
    
    }



    return true;
}

// hipDeviceSynchronize();
// print_GPU<<<1,1>>>( d_res0 );
// printVector_GPU<<<1,18>>>( d_c, 18 );

   
// hipDeviceSynchronize();
// printVector_GPU<<< 1, 1 >>>( m_d_res0,1 );
// hipDeviceSynchronize();
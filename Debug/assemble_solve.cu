#include "hip/hip_runtime.h"
/*
    
*/

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime.h>
// #include "../include/mycudaheader.h"
// #include "precond.h"
#include "cudakernels.h"
#include "assemble.h"
#include "solver.h"
#include "tdo.h"

using namespace std;

// TODO: matrix assembly 2D 3D
// TODO: store local k matrix in constant memory
// TODO: fix prolongation assembly - has something to do with bc initialization
// TODO: 3d elements' node distribution
// TODO: bc cases
// TODO: work on 2D with 3 levels
// TODO: create R matrix = transpose P



int main()
{

    // Material properties
    double youngMod = 210e6;
    double poisson = 0.3;

    // domain dimensions (x,y,z) on coarsest grid
    vector<size_t> N = {1,1};
    // vector<size_t> N = {1,1,1};
    size_t dim = N.size();

    // multigrid precond
    size_t numLevels = 2;
    
    // local element mesh size on coarsest grid
    double h_coarse = 0.5;
    
    // calculating the mesh size on the top level grid
    double h = h_coarse/pow(2,numLevels - 1);

    // smoother (jacobi damping parameter)
    double damp = 2.0/3.0;
    
    // boundary conditions (nodes)
    // TODO: give BC cases
        // MBB with fixed sides
        // MBB with ...
    // TODO: assembleBC( size_t case );

    vector<vector<size_t>> bc_index(numLevels);

    // 1x1 base
        bc_index[0] = {0, 2};
        bc_index[1] = {0, 3, 6};
        // bc_index[2] = {0, 5, 10, 15, 20};

    // bc_index[0] = {0,5};
    // bc_index[1] = {0,9,18};
    // bc_index[2] = {0,17,34,51,68};
   
    // TDO
    double rho = 0.4;
    size_t p = 3;

    vector<size_t> num_rows;
    vector<size_t> max_row_size;
    vector<size_t> p_max_row_size;
    vector<size_t> r_max_row_size;

    //// device pointers

    // local stiffness
    double* d_A_local;

    // global stiffness matrix on each grid-level
    vector<double*> d_value;
    vector<size_t*> d_index;

    // prolongation matrices
    vector<double*> d_p_value;
    vector<size_t*> d_p_index;

    // restriction matrices
    vector<double*> d_r_value;
    vector<size_t*> d_r_index;

    // design variable
    double* d_kai;           // NOTE: can alloc this immediately



    //// CUDA
    vector<size_t*> d_node_index;
    // d_node_index.resize(4);

    /*
    ##################################################################
    #                           ASSEMBLY                             #
    ##################################################################
    */
    
    

    Assembler Assembly(dim, h, N, youngMod, poisson, rho, p, numLevels);
    Assembly.setBC(bc_index);
    Assembly.init(d_A_local, d_value, d_index, d_p_value, d_p_index, d_r_value, d_r_index, d_kai, num_rows, max_row_size, p_max_row_size, r_max_row_size, d_node_index);

    /*
    NOTE: after assembling you should have these :
    global stiffness matrix ELLPACK
        - vector<double*> d_value(numLevels)
        - vector<size_t> d_index(numLevels)
        - vector<size_t> max_row_size(numLevels)
        - vector<double*> d_p_value(numLevels - 1)
        - vector<size_t*> d_p_index(numLevels - 1)
        - vector<size_t> p_max_row_size(numLevels -1 )
    */
   
    // vector u, b
    vector<double> b(num_rows[numLevels - 1], 0);
    b[5] = -10000;  // 1x1 base, 2 levels
    // b[9] = -10000;  // 1x1 base, 3 levels
    // b[105] = -10000;
    // b[49] = -10000;
    double* d_u;
    double* d_b;

    // TODO: get num_rows
    CUDA_CALL( hipMalloc((void**)&d_u, sizeof(double) * num_rows[numLevels - 1] ) );
    CUDA_CALL( hipMalloc((void**)&d_b, sizeof(double) * num_rows[numLevels - 1] ) );

    CUDA_CALL( hipMemset(d_u, 0, sizeof(double) * num_rows[numLevels - 1]) );

    CUDA_CALL( hipMemcpy(d_b, &b[0], sizeof(double) * num_rows[numLevels - 1], hipMemcpyHostToDevice) );

    /*
    ##################################################################
    #                           SOLVER                               #
    ##################################################################
    */

    // TODO: remove num_cols
        

    // printELL_GPU<<<1,1>>> ( d_value[0], d_index[0], max_row_size[0], num_rows[0], num_rows[0]);
    // printELL_GPU<<<1,1>>> ( d_value[1], d_index[1], max_row_size[1], num_rows[1], num_rows[1]);
    // printELL_GPU<<<1,1>>> ( d_value[2], d_index[2], max_row_size[2], num_rows[2], num_rows[2]);



    Solver GMG(d_value, d_index, d_p_value, d_p_index, numLevels, num_rows, max_row_size, p_max_row_size, damp);

    GMG.init();
    GMG.set_verbose(false, false);
    GMG.set_num_prepostsmooth(3,3);
    GMG.set_convergence_params(1, 1e-99, 1e-10);
    GMG.set_bs_convergence_params(1, 1e-99, 1e-10);
    GMG.set_cycle('V');
    GMG.set_steps(15, 5); 
    hipDeviceSynchronize();
    GMG.solve(d_u, d_b, d_value);
    hipDeviceSynchronize();

    // hipDeviceSynchronize();

    // printVector_GPU<<<1,num_rows[numLevels - 1]>>>( d_u, num_rows[numLevels - 1]);
    
    
    // /*
    // ##################################################################
    // #                           TDO                                  #
    // ##################################################################
    // */
    
    // // // TDO algorithm, tdo.cu
    // // // produces updated d_kai
    // // cout << "\n";
    // // cout << "TDO" << endl;

    // printVector_GPU<<<1,1>>>( d_A_local, 1);

    // TODO: incorporate this in the beginning
    double etastar = 12.0;
    double betastar = 2.0 * pow(h,2);

    // TODO: p=3 is not incorporated
    TDO tdo(d_u, d_kai, h, dim, betastar, etastar, Assembly.getNumElements(), num_rows[0], d_A_local, d_node_index, N, rho);
    tdo.init();
    tdo.innerloop();    // get updated d_kai

    hipDeviceSynchronize();
//     // // DEBUG:
        // printVector_GPU<<<1,Assembly.getNumElements()>>>( d_kai, Assembly.getNumElements());

    // // update stiffness matrix with new d_kai
    // // TODO: get d_value, d_index and d_A_local from the class, 
    // // in the end, it's only Update..(d_kai)


    Assembly.UpdateGlobalStiffness(d_kai, d_value, d_index, d_p_value, d_p_index, d_r_value, d_r_index, d_A_local);
    // hipDeviceSynchronize();    

    // printELL_GPU<<<1,1>>> ( d_value[0], d_index[0], max_row_size[0], num_rows[0], num_rows[0]);
    printELL_GPU<<<1,1>>> ( d_value[1], d_index[1], max_row_size[1], num_rows[1], num_rows[1]);
    // printVector_GPU<<<1,1>>>( d_value[1], 1 );
    hipDeviceSynchronize();
//     // printELL_GPU<<<1,1>>> ( d_p_value[0], d_p_index[0], p_max_row_size[0], num_rows[1], num_rows[0]);
//     // printELL_GPU<<<1,1>>> ( d_r_value[0], d_r_index[0], r_max_row_size[0], num_rows[0], num_rows[1]);


//     // GMG.reinit(); // TODO: update global matrix here, update the coarser ones here too 
//     hipDeviceSynchronize();

    
// //     // TODO: remove d_value here
// //     GMG.solve(d_u, d_b, d_value);


//     temp<<<1,4>>>(d_kai);

    
    
    // PTAP_GPU consider using 2d blocks? :
    // https://www.quantstart.com/articles/Matrix-Matrix-Multiplication-on-the-GPU-with-Nvidia-CUDA/



}


// print_GPU<<<1,1>>> ( d_res0 );
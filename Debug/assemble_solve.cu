#include "hip/hip_runtime.h"
/*
    
*/

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_runtime.h>
// #include "../include/mycudaheader.h"
// #include "precond.h"
#include "cudakernels.h"
#include "solver.h"

using namespace std;


int main()
{
    size_t num_rows = 18;
    size_t num_cols = 18;

    size_t max_row_size;
    size_t N = 2;

    vector<size_t> bc_index = {0, 1, 6, 7, 12, 13};

    // number of levels in GMG
    size_t numLevels = 2;


    // displacement vector
    vector<double> u(num_rows);
    double* d_u;

    // force vector
    vector<double> b(num_rows);
    double* d_b;

    // add forces
    b[5] = -10;

    // residuum vector
    double* d_r;

    // correction vector
    double* d_c;

    vector<double> A_g = {
        6652102.4,	2400134.4,	-4066334.72,	-185606.4,	0,	0,	740236.8,	185606.4,	-3325952,	-2400153.6,	0,	0,	0,	0,	0,	0,	0,	0,
        2400134.4,	6652102.4,	185606.4,	740236.8,	0,	0,	-185606.4,	-4066334.72,	-2400153.6,	-3325952,	0,	0,	0,	0,	0,	0,	0,	0,
        -4066334.72,	185606.4,	13304204.8,	0,	-4066334.72,	-185606.4,	-3325952,	2400153.6,	1480473.6,	0,	-3325952,	-2400153.6,	0,	0,	0,	0,	0,	0,
        -185606.4,	740236.8,	0,	13304204.8,	185606.4,	740236.8,	2400153.6,	-3325952,	0,	-8132669.44,	-2400153.6,	-3325952,	0,	0,	0,	0,	0,	0,
        0,	0,	-4066334.72,	185606.4,	6652102.4,	-2400134.4,	0,	0,	-3325952,	2400153.6,	740236.8,	-185606.4,	0,	0,	0,	0,	0,	0,
        0,	0,	-185606.4,	740236.8,	-2400134.4,	6652102.4,	0,	0,	2400153.6,	-3325952,	185606.4,	-4066334.72,	0,	0,	0,	0,	0,	0,
        740236.8,	-185606.4,	-3325952,	2400153.6,	0,	0,	13304204.8,	0,	-8132669.44,	0,	0,	0,	740236.8,	185606.4,	-3325952,	-2400153.6,	0,	0,
        185606.4,	-4066334.72,	2400153.6,	-3325952,	0,	0,	0,	13304204.8,	0,	1480473.6,	0,	0,	-185606.4,	-4066334.72,	-2400153.6,	-3325952,	0,	0,
        -3325952,	-2400153.6,	1480473.6,	0,	-3325952,	2400153.6,	-8132669.44,	0,	26608409.6,	0,	-8132669.44,	0,	-3325952,	2400153.6,	1480473.6,	0,	-3325952,	-2400153.6,
        -2400153.6,	-3325952,	0,	-8132669.44,	2400153.6,	-3325952,	0,	1480473.6,	0,	26608409.6,	0,	1480473.6,	2400153.6,	-3325952,	0,	-8132669.44,	-2400153.6,	-3325952,
        0,	0,	-3325952,	-2400153.6,	740236.8,	185606.4,	0,	0,	-8132669.44,	0,	13304204.8,	0,	0,	0,	-3325952,	2400153.6,	740236.8,	-185606.4,
        0,	0,	-2400153.6,	-3325952,	-185606.4,	-4066334.72,	0,	0,	0,	1480473.6,	0,	13304204.8,	0,	0,	2400153.6,	-3325952,	185606.4,	-4066334.72,
        0,	0,	0,	0,	0,	0,	740236.8,	-185606.4,	-3325952,	2400153.6,	0,	0,	6652102.4,	-2400134.4,	-4066334.72,	185606.4,	0,	0,
        0,	0,	0,	0,	0,	0,	185606.4,	-4066334.72,	2400153.6,	-3325952,	0,	0,	-2400134.4,	6652102.4,	-185606.4,	740236.8,	0,	0,
        0,	0,	0,	0,	0,	0,	-3325952,	-2400153.6,	1480473.6,	0,	-3325952,	2400153.6,	-4066334.72,	-185606.4,	13304204.8,	0,	-4066334.72,	185606.4,
        0,	0,	0,	0,	0,	0,	-2400153.6,	-3325952,	0,	-8132669.44,	2400153.6,	-3325952,	185606.4,	740236.8,	0,	13304204.8,	-185606.4,	740236.8,
        0,	0,	0,	0,	0,	0,	0,	0,	-3325952,	-2400153.6,	740236.8,	185606.4,	0,	0,	-4066334.72,	-185606.4,	6652102.4,	2400134.4,
        0,	0,	0,	0,	0,	0,	0,	0,	-2400153.6,	-3325952,	-185606.4,	-4066334.72,	0,	0,	185606.4,	740236.8,	2400134.4,	6652102.4
    };

    std::vector<double> value;
    std::vector<std::size_t> index;

    double *d_value;
    size_t *d_index;

    for ( int i = 0 ; i < bc_index.size() ; ++i )
        applyMatrixBC(&A_g[0], bc_index[i], num_rows, num_cols);

    // get max row size
    max_row_size = getMaxRowSize(A_g, num_rows, num_cols);

    value.resize(max_row_size*num_rows, 0.0);
    index.resize(max_row_size*num_rows, 0);

    transformToELL(A_g, value, index, max_row_size, num_rows);

    // cuda
    CUDA_CALL( hipMalloc((void**)&d_u, sizeof(double) * num_rows) );
    CUDA_CALL( hipMalloc((void**)&d_b, sizeof(double) * num_rows) );
    CUDA_CALL( hipMalloc((void**)&d_r, sizeof(double) * num_rows) );
    CUDA_CALL( hipMalloc((void**)&d_c, sizeof(double) * num_rows) );
    CUDA_CALL( hipMalloc((void**)&d_value, sizeof(double) * max_row_size*num_rows) );
    CUDA_CALL( hipMalloc((void**)&d_index, sizeof(size_t) * max_row_size*num_rows) );

    CUDA_CALL( hipMemset(d_u, 0, sizeof(double) * num_rows) );
    CUDA_CALL( hipMemset(d_r, 0, sizeof(double) * num_rows) );
    CUDA_CALL( hipMemset(d_c, 0, sizeof(double) * num_rows) );
    

    CUDA_CALL( hipMemcpy(d_value, &value[0], sizeof(double) * max_row_size*num_rows, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(d_index, &index[0], sizeof(size_t) * max_row_size*num_rows, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(d_b, &b[0], sizeof(double) * num_rows, hipMemcpyHostToDevice) );

    dim3 gridDim;
    dim3 blockDim;
    
    // Calculating the required CUDA grid and block dimensions
    calculateDimensions(num_rows, gridDim, blockDim);


    /*
    ##################################################################
    #                           SOLVER                               #
    ##################################################################
    */


    Solver GMG(d_value, d_index, max_row_size, d_u, d_b, 2, num_rows, num_cols);

    GMG.init();
    GMG.set_num_presmooth(3);
    GMG.set_num_postsmooth(3);

    hipDeviceSynchronize();
    GMG.solve(d_u, d_b);

    hipDeviceSynchronize();

    
}


// print_GPU<<<1,1>>> ( d_res0 );